#include "hip/hip_runtime.h"
/*
 * Copyright (C) 2023, Inria
 * GRAPHDECO research group, https://team.inria.fr/graphdeco
 * All rights reserved.
 *
 * This software is free for non-commercial, research and evaluation use 
 * under the terms of the LICENSE.md file.
 *
 * For inquiries contact  george.drettakis@inria.fr
 */

#include "backward.h"
#include "auxiliary.h"
#include <hip/hip_cooperative_groups.h>
#include <cooperative_groups/reduce.h>
#include <cmath>
namespace cg = cooperative_groups;

// Backward pass for conversion of spherical harmonics to RGB for
// each Gaussian.
__device__ void computeColorFromSH(int idx, int deg, int max_coeffs, const glm::vec3* means, glm::vec3 campos, const float* shs, const bool* clamped, const glm::vec3* dL_dcolor, glm::vec3* dL_dmeans, glm::vec3* dL_dshs)
{
	// Compute intermediate values, as it is done during forward
	glm::vec3 pos = means[idx];
	glm::vec3 dir_orig = pos - campos;
	glm::vec3 dir = dir_orig / glm::length(dir_orig);

	glm::vec3* sh = ((glm::vec3*)shs) + idx * max_coeffs;

	// Use PyTorch rule for clamping: if clamping was applied,
	// gradient becomes 0.
	glm::vec3 dL_dRGB = dL_dcolor[idx];
	dL_dRGB.x *= clamped[3 * idx + 0] ? 0 : 1;
	dL_dRGB.y *= clamped[3 * idx + 1] ? 0 : 1;
	dL_dRGB.z *= clamped[3 * idx + 2] ? 0 : 1;

	glm::vec3 dRGBdx(0, 0, 0);
	glm::vec3 dRGBdy(0, 0, 0);
	glm::vec3 dRGBdz(0, 0, 0);
	float x = dir.x;
	float y = dir.y;
	float z = dir.z;

	// Target location for this Gaussian to write SH gradients to
	glm::vec3* dL_dsh = dL_dshs + idx * max_coeffs;

	// No tricks here, just high school-level calculus.
	float dRGBdsh0 = SH_C0;
	dL_dsh[0] = dRGBdsh0 * dL_dRGB;
	if (deg > 0)
	{
		float dRGBdsh1 = -SH_C1 * y;
		float dRGBdsh2 = SH_C1 * z;
		float dRGBdsh3 = -SH_C1 * x;
		dL_dsh[1] = dRGBdsh1 * dL_dRGB;
		dL_dsh[2] = dRGBdsh2 * dL_dRGB;
		dL_dsh[3] = dRGBdsh3 * dL_dRGB;

		dRGBdx = -SH_C1 * sh[3];
		dRGBdy = -SH_C1 * sh[1];
		dRGBdz = SH_C1 * sh[2];

		if (deg > 1)
		{
			float xx = x * x, yy = y * y, zz = z * z;
			float xy = x * y, yz = y * z, xz = x * z;

			float dRGBdsh4 = SH_C2[0] * xy;
			float dRGBdsh5 = SH_C2[1] * yz;
			float dRGBdsh6 = SH_C2[2] * (2.f * zz - xx - yy);
			float dRGBdsh7 = SH_C2[3] * xz;
			float dRGBdsh8 = SH_C2[4] * (xx - yy);
			dL_dsh[4] = dRGBdsh4 * dL_dRGB;
			dL_dsh[5] = dRGBdsh5 * dL_dRGB;
			dL_dsh[6] = dRGBdsh6 * dL_dRGB;
			dL_dsh[7] = dRGBdsh7 * dL_dRGB;
			dL_dsh[8] = dRGBdsh8 * dL_dRGB;

			dRGBdx += SH_C2[0] * y * sh[4] + SH_C2[2] * 2.f * -x * sh[6] + SH_C2[3] * z * sh[7] + SH_C2[4] * 2.f * x * sh[8];
			dRGBdy += SH_C2[0] * x * sh[4] + SH_C2[1] * z * sh[5] + SH_C2[2] * 2.f * -y * sh[6] + SH_C2[4] * 2.f * -y * sh[8];
			dRGBdz += SH_C2[1] * y * sh[5] + SH_C2[2] * 2.f * 2.f * z * sh[6] + SH_C2[3] * x * sh[7];

			if (deg > 2)
			{
				float dRGBdsh9 = SH_C3[0] * y * (3.f * xx - yy);
				float dRGBdsh10 = SH_C3[1] * xy * z;
				float dRGBdsh11 = SH_C3[2] * y * (4.f * zz - xx - yy);
				float dRGBdsh12 = SH_C3[3] * z * (2.f * zz - 3.f * xx - 3.f * yy);
				float dRGBdsh13 = SH_C3[4] * x * (4.f * zz - xx - yy);
				float dRGBdsh14 = SH_C3[5] * z * (xx - yy);
				float dRGBdsh15 = SH_C3[6] * x * (xx - 3.f * yy);
				dL_dsh[9] = dRGBdsh9 * dL_dRGB;
				dL_dsh[10] = dRGBdsh10 * dL_dRGB;
				dL_dsh[11] = dRGBdsh11 * dL_dRGB;
				dL_dsh[12] = dRGBdsh12 * dL_dRGB;
				dL_dsh[13] = dRGBdsh13 * dL_dRGB;
				dL_dsh[14] = dRGBdsh14 * dL_dRGB;
				dL_dsh[15] = dRGBdsh15 * dL_dRGB;

				dRGBdx += (
					SH_C3[0] * sh[9] * 3.f * 2.f * xy +
					SH_C3[1] * sh[10] * yz +
					SH_C3[2] * sh[11] * -2.f * xy +
					SH_C3[3] * sh[12] * -3.f * 2.f * xz +
					SH_C3[4] * sh[13] * (-3.f * xx + 4.f * zz - yy) +
					SH_C3[5] * sh[14] * 2.f * xz +
					SH_C3[6] * sh[15] * 3.f * (xx - yy));

				dRGBdy += (
					SH_C3[0] * sh[9] * 3.f * (xx - yy) +
					SH_C3[1] * sh[10] * xz +
					SH_C3[2] * sh[11] * (-3.f * yy + 4.f * zz - xx) +
					SH_C3[3] * sh[12] * -3.f * 2.f * yz +
					SH_C3[4] * sh[13] * -2.f * xy +
					SH_C3[5] * sh[14] * -2.f * yz +
					SH_C3[6] * sh[15] * -3.f * 2.f * xy);

				dRGBdz += (
					SH_C3[1] * sh[10] * xy +
					SH_C3[2] * sh[11] * 4.f * 2.f * yz +
					SH_C3[3] * sh[12] * 3.f * (2.f * zz - xx - yy) +
					SH_C3[4] * sh[13] * 4.f * 2.f * xz +
					SH_C3[5] * sh[14] * (xx - yy));
			}
		}
	}

	// The view direction is an input to the computation. View direction
	// is influenced by the Gaussian's mean, so SHs gradients
	// must propagate back into 3D position.
	glm::vec3 dL_ddir(glm::dot(dRGBdx, dL_dRGB), glm::dot(dRGBdy, dL_dRGB), glm::dot(dRGBdz, dL_dRGB));

	// Account for normalization of direction
	float3 dL_dmean = dnormvdv(float3{ dir_orig.x, dir_orig.y, dir_orig.z }, float3{ dL_ddir.x, dL_ddir.y, dL_ddir.z });

	// Gradients of loss w.r.t. Gaussian means, but only the portion 
	// that is caused because the mean affects the view-dependent color.
	// Additional mean gradient is accumulated in below methods.
	dL_dmeans[idx] += glm::vec3(dL_dmean.x, dL_dmean.y, dL_dmean.z);
}

// Backward version of INVERSE 2D covariance matrix computation
// (due to length launched as separate kernel before other 
// backward steps contained in preprocess)
__global__ void computeCov2DCUDA(int P,
	const float3* means,
	const int* radii,
	const float* cov3Ds,
	const float h_x, float h_y,
	const float tan_fovx, float tan_fovy,
	const float* view_matrix,
	const float* dL_dconics,
	const float2* dL_depths_plane,
	const float3* dL_dnormals,
	float3* dL_dmeans,
	float* dL_dcov,
	const float4* __restrict__ conic_opacity,
	float* dL_dopacity)
{
	auto idx = cg::this_grid().thread_rank();
	if (idx >= P || !(radii[idx] > 0))
		return;

	// Reading location of 3D covariance for this Gaussian
	const float* cov3D = cov3Ds + 6 * idx;

	// Fetch gradients, recompute 2D covariance and relevant 
	// intermediate forward results needed in the backward.
	float3 mean = means[idx];
	float3 dL_dconic = { dL_dconics[4 * idx], dL_dconics[4 * idx + 1], dL_dconics[4 * idx + 3] };
	const glm::vec3 dL_dnormal = { dL_dnormals[idx].x,dL_dnormals[idx].y,dL_dnormals[idx].z};
	const float4 conic = conic_opacity[idx];
	const float combined_opacity = conic.w;
	const float2& dL_depth_plane_combined = dL_depths_plane[idx];
	// const float2& dL_depth_plane = dL_depths_plane[idx];
	// printf("%.10f %.10f \n",dL_depth_plane.x,dL_depth_plane.y);
	float3 t = transformPoint4x3(mean, view_matrix);
	
	const float limx = 1.3f * tan_fovx;
	const float limy = 1.3f * tan_fovy;
	float txtz = t.x / t.z;
	float tytz = t.y / t.z;
	t.x = min(limx, max(-limx, txtz)) * t.z;
	t.y = min(limy, max(-limy, tytz)) * t.z;
	
	const float x_grad_mul = txtz < -limx || txtz > limx ? 0 : 1;
	const float y_grad_mul = tytz < -limy || tytz > limy ? 0 : 1;

	txtz = t.x / t.z;
	tytz = t.y / t.z;

	glm::mat3 J = glm::mat3(h_x / t.z, 0.0f, -(h_x * t.x) / (t.z * t.z),
		0.0f, h_y / t.z, -(h_y * t.y) / (t.z * t.z),
		0, 0, 0);

	glm::mat3 W = glm::mat3(
		view_matrix[0], view_matrix[4], view_matrix[8],
		view_matrix[1], view_matrix[5], view_matrix[9],
		view_matrix[2], view_matrix[6], view_matrix[10]);

	glm::mat3 Vrk = glm::mat3(
		cov3D[0], cov3D[1], cov3D[2],
		cov3D[1], cov3D[3], cov3D[4],
		cov3D[2], cov3D[4], cov3D[5]);

	glm::mat3 T = W * J;

	glm::mat3 cov2D = glm::transpose(T) * glm::transpose(Vrk) * T;

	const float det_0 = max(1e-6, cov2D[0][0] * cov2D[1][1] - cov2D[0][1] * cov2D[0][1]);
	const float det_1 = max(1e-6, (cov2D[0][0] + kernel_size) * (cov2D[1][1] + kernel_size) - cov2D[0][1] * cov2D[0][1]);
	// sqrt here
	const float coef = sqrt(det_0 / (det_1+1e-6) + 1e-6);
	// const float coef = 1.0f;

	glm::mat3 Vrk_eigen_vector;
	glm::vec3 Vrk_eigen_value;
	int D = glm::findEigenvaluesSymReal(Vrk,Vrk_eigen_value,Vrk_eigen_vector);

	unsigned int min_id = Vrk_eigen_value[0]>Vrk_eigen_value[1]? (Vrk_eigen_value[1]>Vrk_eigen_value[2]?2:1):(Vrk_eigen_value[0]>Vrk_eigen_value[2]?2:0);

	glm::mat3 Vrk_inv;
	glm::vec3 eigenvector_min;
	bool well_conditioned = Vrk_eigen_value[min_id]>0.00000001;
	if(well_conditioned)
	{
		glm::mat3 diag = glm::mat3(1/Vrk_eigen_value[0],0,0,
									0,1/Vrk_eigen_value[1],0,
									0,0,1/Vrk_eigen_value[2]);
		Vrk_inv = Vrk_eigen_vector * diag * glm::transpose(Vrk_eigen_vector);
	}
	else
	{
		if(D<3)
		{
			const glm::vec3 eigenvector1 = Vrk_eigen_vector[(min_id+1)%3];
			const glm::vec3 eigenvector2 = Vrk_eigen_vector[(min_id+2)%3];
			eigenvector_min = glm::cross(eigenvector1, eigenvector2);
		}
		else{
			eigenvector_min = Vrk_eigen_vector[min_id];
		}
		Vrk_inv = glm::outerProduct(eigenvector_min,eigenvector_min);
	}
	
	// glm::mat3 Vrk_inv = Vrk_eigen_vector * diag * glm::transpose(Vrk_eigen_vector);
	glm::mat3 cov_cam_inv = glm::transpose(W) * Vrk_inv * W;
	glm::vec3 uvh = {txtz, tytz, 1};
	glm::vec3 uvh_m = cov_cam_inv * uvh;
	glm::vec3 uvh_mn = glm::normalize(uvh_m);
	
	float vb = glm::dot(uvh_m, uvh);
	float vbn = glm::dot(uvh_mn, uvh);
	float u2 = txtz * txtz;
	float v2 = tytz * tytz;
	float uv = txtz * tytz;

	float l = sqrt(t.x*t.x+t.y*t.y+t.z*t.z);
	glm::mat3 Jn = glm::mat3(
		1 / t.z, 0.0f, -(t.x) / (t.z * t.z),
		0.0f, 1 / t.z, -(t.y) / (t.z * t.z),
		t.x/l, t.y/l, t.z/l);

	glm::mat3 nJ_inv = glm::mat3(
		v2 + 1,	-uv, 		0,
		-uv,	u2 + 1,		0,
		-txtz,	-tytz,		0
	);

	float clamp_vb = max(vb, 0.0000001f);
	float clamp_vbn = max(vbn, 0.0000001f);
	float factor = t.z / ((u2+v2+1) * clamp_vb);
	float factor2 = t.z / (u2+v2+1);
	float factor_normal = l / (u2+v2+1);
	glm::vec3 uvh_m_vb = uvh_mn/clamp_vbn;
	glm::vec3 planen = nJ_inv * uvh_m_vb;
	float2 depth_plane = { planen[0]*factor2, planen[1]*factor2};

	glm::vec3 ray_normal_vector = {-planen[0]*factor_normal, -planen[1]*factor_normal, -1};

	glm::vec3 cam_normal_vector = Jn * ray_normal_vector;
	glm::vec3 normal_vector = glm::normalize(cam_normal_vector);
	float lv2 = (cam_normal_vector.x*cam_normal_vector.x+\
				cam_normal_vector.y*cam_normal_vector.y+\
				cam_normal_vector.z*cam_normal_vector.z);
	float lv = sqrt(lv2);
	float lv3 = lv2 * lv;
	glm::mat3 eye3 = glm::mat3(1,0,0,
								0,1,0,
								0,0,1);
	const glm::vec3 dL_dnormal_lv = dL_dnormal/lv;
	glm::vec3 dL_dcam_normal_vector = dL_dnormal_lv - normal_vector * glm::dot(normal_vector,dL_dnormal_lv);
	glm::vec3 dL_dray_normal_vector = glm::transpose(Jn) * dL_dcam_normal_vector;
	const float l_z = l / t.z;
	const float dL_dl_z = - (dL_dray_normal_vector.x * depth_plane.x + dL_dray_normal_vector.y * depth_plane.y);
	const glm::vec2 dL_depth_plane = {dL_depth_plane_combined.x - dL_dray_normal_vector.x * l_z, dL_depth_plane_combined.y - dL_dray_normal_vector.y * l_z};
	glm::mat3 dL_dJn = glm::outerProduct(dL_dcam_normal_vector,ray_normal_vector);

	float tmp = dL_depth_plane.x * depth_plane.x + dL_depth_plane.y * depth_plane.y;
	float dL_dvb = -tmp / clamp_vb;

	glm::vec3 dL_dplane = {dL_depth_plane.x * (factor), dL_depth_plane.y * (factor), 0};

	

	glm::vec3 W_uvh = W * uvh;
	glm::vec3 nJ_inv_dL_dplane = glm::transpose(nJ_inv) * dL_dplane;
	glm::mat3 dL_dVrk = glm::mat3();

	if(well_conditioned){
		dL_dVrk = - glm::outerProduct(Vrk_inv * W_uvh, (Vrk_inv/clamp_vb) * (W_uvh * (-tmp) + W * glm::transpose(nJ_inv) * glm::vec3(dL_depth_plane.x * factor2, dL_depth_plane.y * factor2, 0)));
	}
	else{
		dL_dVrk = glm::mat3(0,0,0,0,0,0,0,0,0);
		glm::mat3 dL_dVrk_inv = glm::outerProduct(W_uvh, W_uvh * dL_dvb + W * nJ_inv_dL_dplane);
		glm::vec3 dL_dv = (dL_dVrk_inv + glm::transpose(dL_dVrk_inv)) * eigenvector_min;;
		for(int j =0;j<3;j++)
		{
			if(j!=min_id)
			{
				float scale = glm::dot(Vrk_eigen_vector[j],dL_dv)/min(Vrk_eigen_value[min_id] - Vrk_eigen_value[j], - 0.0000001f);
				dL_dVrk += glm::outerProduct(Vrk_eigen_vector[j] * scale, eigenvector_min);
			}
		}
	}
	
	
	glm::vec3 vec_dL_depth_plane = glm::vec3(dL_depth_plane.x,dL_depth_plane.y,0);
	glm::vec3 dL_duvh = 2 * (-tmp) * uvh_m_vb + (cov_cam_inv/clamp_vb) * glm::transpose(nJ_inv) * glm::vec3(dL_depth_plane.x * factor2, dL_depth_plane.y * factor2, 0);
	
	glm::mat3 dL_dnJ_inv = glm::outerProduct(vec_dL_depth_plane, uvh_m_vb * factor2);

	float dL_du = -tmp * 2 * txtz / (u2+v2+1)
					+ dL_duvh.x 
					+ (dL_dnJ_inv[0][1] + dL_dnJ_inv[1][0]) * (-tytz) + 2 * dL_dnJ_inv[1][1] * txtz - dL_dnJ_inv[2][0]
					+ dL_dl_z * txtz / sqrt(u2+v2+1); //this line is from normal
	float dL_dv = -tmp * 2 * tytz / (u2+v2+1)
					+ dL_duvh.y 
					+ (dL_dnJ_inv[0][1] + dL_dnJ_inv[1][0]) * (-txtz) + 2 * dL_dnJ_inv[0][0] * tytz - dL_dnJ_inv[2][1]
					+ dL_dl_z * tytz / sqrt(u2+v2+1); //this line is from normal


	const float opacity = combined_opacity / (coef + 1e-6);
	const float dL_dcoef = dL_dopacity[idx] * opacity;
	const float dL_dsqrtcoef = dL_dcoef * 0.5 * 1. / (coef + 1e-6);
	const float dL_ddet0 = dL_dsqrtcoef / (det_1+1e-6);
	const float dL_ddet1 = dL_dsqrtcoef * det_0 * (-1.f / (det_1 * det_1 + 1e-6));
	//TODO gradient is zero if det_0 or det_1 < 0
	const float dcoef_da = dL_ddet0 * cov2D[1][1] + dL_ddet1 * (cov2D[1][1] + kernel_size);
	const float dcoef_db = dL_ddet0 * (-2. * cov2D[0][1]) + dL_ddet1 * (-2. * cov2D[0][1]);
	const float dcoef_dc = dL_ddet0 * cov2D[0][0] + dL_ddet1 * (cov2D[0][0] + kernel_size);
	// Use helper variables for 2D covariance entries. More compact.
	float a = cov2D[0][0] + kernel_size;
	float b = cov2D[0][1];
	float c = cov2D[1][1] + kernel_size;

	float denom = a * c - b * b;
	float dL_da = 0, dL_db = 0, dL_dc = 0;
	float denom2inv = 1.0f / ((denom * denom) + 0.0000001f);

	if (denom2inv != 0)
	{
		// Gradients of loss w.r.t. entries of 2D covariance matrix,
		// given gradients of loss w.r.t. conic matrix (inverse covariance matrix).
		// e.g., dL / da = dL / d_conic_a * d_conic_a / d_a
		dL_da = denom2inv * (-c * c * dL_dconic.x + 2 * b * c * dL_dconic.y + (denom - a * c) * dL_dconic.z);
		dL_dc = denom2inv * (-a * a * dL_dconic.z + 2 * a * b * dL_dconic.y + (denom - a * c) * dL_dconic.x);
		dL_db = denom2inv * 2 * (b * c * dL_dconic.x - (denom + 2 * b * b) * dL_dconic.y + a * b * dL_dconic.z);

		if (det_0 <= 1e-6 || det_1 <= 1e-6){
			dL_dopacity[idx] = 0;
		} else {
			// Gradiends of alpha respect to conv due to low pass filter
			dL_da += dcoef_da;
			dL_dc += dcoef_dc;
			dL_db += dcoef_db;

			// update dL_dopacity
			dL_dopacity[idx] = dL_dopacity[idx] * coef;
		}

		// Gradients of loss L w.r.t. each 3D covariance matrix (Vrk) entry, 
		// given gradients w.r.t. 2D covariance matrix (diagonal).
		// cov2D = transpose(T) * transpose(Vrk) * T;
		dL_dcov[6 * idx + 0] = (T[0][0] * T[0][0] * dL_da + T[0][0] * T[1][0] * dL_db + T[1][0] * T[1][0] * dL_dc);
		dL_dcov[6 * idx + 3] = (T[0][1] * T[0][1] * dL_da + T[0][1] * T[1][1] * dL_db + T[1][1] * T[1][1] * dL_dc);
		dL_dcov[6 * idx + 5] = (T[0][2] * T[0][2] * dL_da + T[0][2] * T[1][2] * dL_db + T[1][2] * T[1][2] * dL_dc);

		// Gradients of loss L w.r.t. each 3D covariance matrix (Vrk) entry, 
		// given gradients w.r.t. 2D covariance matrix (off-diagonal).
		// Off-diagonal elements appear twice --> double the gradient.
		// cov2D = transpose(T) * transpose(Vrk) * T;
		dL_dcov[6 * idx + 1] = 2 * T[0][0] * T[0][1] * dL_da + (T[0][0] * T[1][1] + T[0][1] * T[1][0]) * dL_db + 2 * T[1][0] * T[1][1] * dL_dc;
		dL_dcov[6 * idx + 2] = 2 * T[0][0] * T[0][2] * dL_da + (T[0][0] * T[1][2] + T[0][2] * T[1][0]) * dL_db + 2 * T[1][0] * T[1][2] * dL_dc;
		dL_dcov[6 * idx + 4] = 2 * T[0][2] * T[0][1] * dL_da + (T[0][1] * T[1][2] + T[0][2] * T[1][1]) * dL_db + 2 * T[1][1] * T[1][2] * dL_dc;
	}
	else
	{
		for (int i = 0; i < 6; i++)
			dL_dcov[6 * idx + i] = 0;
	}
	dL_dcov[6 * idx + 0] += dL_dVrk[0][0];
	dL_dcov[6 * idx + 3] += dL_dVrk[1][1];
	dL_dcov[6 * idx + 5] += dL_dVrk[2][2];
	dL_dcov[6 * idx + 1] += dL_dVrk[0][1] + dL_dVrk[1][0];
	dL_dcov[6 * idx + 2] += dL_dVrk[0][2] + dL_dVrk[2][0];
	dL_dcov[6 * idx + 4] += dL_dVrk[1][2] + dL_dVrk[2][1];


	// Gradients of loss w.r.t. upper 2x3 portion of intermediate matrix T
	// cov2D = transpose(T) * transpose(Vrk) * T;
	float dL_dT00 = 2 * (T[0][0] * Vrk[0][0] + T[0][1] * Vrk[0][1] + T[0][2] * Vrk[0][2]) * dL_da +
		(T[1][0] * Vrk[0][0] + T[1][1] * Vrk[0][1] + T[1][2] * Vrk[0][2]) * dL_db;
	float dL_dT01 = 2 * (T[0][0] * Vrk[1][0] + T[0][1] * Vrk[1][1] + T[0][2] * Vrk[1][2]) * dL_da +
		(T[1][0] * Vrk[1][0] + T[1][1] * Vrk[1][1] + T[1][2] * Vrk[1][2]) * dL_db;
	float dL_dT02 = 2 * (T[0][0] * Vrk[2][0] + T[0][1] * Vrk[2][1] + T[0][2] * Vrk[2][2]) * dL_da +
		(T[1][0] * Vrk[2][0] + T[1][1] * Vrk[2][1] + T[1][2] * Vrk[2][2]) * dL_db;
	float dL_dT10 = 2 * (T[1][0] * Vrk[0][0] + T[1][1] * Vrk[0][1] + T[1][2] * Vrk[0][2]) * dL_dc +
		(T[0][0] * Vrk[0][0] + T[0][1] * Vrk[0][1] + T[0][2] * Vrk[0][2]) * dL_db;
	float dL_dT11 = 2 * (T[1][0] * Vrk[1][0] + T[1][1] * Vrk[1][1] + T[1][2] * Vrk[1][2]) * dL_dc +
		(T[0][0] * Vrk[1][0] + T[0][1] * Vrk[1][1] + T[0][2] * Vrk[1][2]) * dL_db;
	float dL_dT12 = 2 * (T[1][0] * Vrk[2][0] + T[1][1] * Vrk[2][1] + T[1][2] * Vrk[2][2]) * dL_dc +
		(T[0][0] * Vrk[2][0] + T[0][1] * Vrk[2][1] + T[0][2] * Vrk[2][2]) * dL_db;

	// Gradients of loss w.r.t. upper 3x2 non-zero entries of Jacobian matrix
	// T = W * J
	float dL_dJ00 = W[0][0] * dL_dT00 + W[0][1] * dL_dT01 + W[0][2] * dL_dT02;
	float dL_dJ02 = W[2][0] * dL_dT00 + W[2][1] * dL_dT01 + W[2][2] * dL_dT02;
	float dL_dJ11 = W[1][0] * dL_dT10 + W[1][1] * dL_dT11 + W[1][2] * dL_dT12;
	float dL_dJ12 = W[2][0] * dL_dT10 + W[2][1] * dL_dT11 + W[2][2] * dL_dT12;

	float tz = 1.f / t.z;
	float tz2 = tz * tz;
	float tz3 = tz2 * tz;


	float dL_dtx = x_grad_mul * (-h_x * tz2 * dL_dJ02 + dL_du * tz
								-dL_dJn[0][2]*tz2 + dL_dJn[2][0]*(1/lv-t.x*t.x/lv3) + dL_dJn[2][2]*(-t.x*t.z/lv3)); //this line is from normal
	float dL_dty = y_grad_mul * (-h_y * tz2 * dL_dJ12 + dL_dv * tz
								-dL_dJn[1][2]*tz2 + dL_dJn[2][1]*(1/lv-t.y*t.y/lv3) + dL_dJn[2][2]*(-t.y*t.z/lv3)); //this line is from normal
	float dL_dtz = -h_x * tz2 * dL_dJ00 - h_y * tz2 * dL_dJ11 + (2 * h_x * t.x) * tz3 * dL_dJ02 + (2 * h_y * t.y) * tz3 * dL_dJ12
					- (dL_du * t.x + dL_dv * t.y) * tz2 + tmp * tz
					+ dL_dJn[0][0] * (-tz2) + dL_dJn[1][1] * (-tz2) + dL_dJn[0][2] * (2*t.x*tz3) + dL_dJn[1][2] * (2*t.y*tz3)
					+ (dL_dJn[2][0]*t.x+dL_dJn[2][1]*t.y)*(-t.z/lv3) + dL_dJn[2][2]*(1/lv-t.z*t.z/lv3); // two lines are from normal


	// Account for transformation of mean to t
	// t = transformPoint4x3(mean, view_matrix);
	float3 dL_dmean = transformVec4x3Transpose({ dL_dtx, dL_dty, dL_dtz }, view_matrix);

	// Gradients of loss w.r.t. Gaussian means, but only the portion 
	// that is caused because the mean affects the covariance matrix.
	// Additional mean gradient is accumulated in BACKWARD::preprocess.
	dL_dmeans[idx] = dL_dmean;
}

// Backward pass for the conversion of scale and rotation to a 
// 3D covariance matrix for each Gaussian. 
__device__ void computeCov3D(int idx, const glm::vec3 scale, float mod, const glm::vec4 rot, const float* dL_dcov3Ds, glm::vec3* dL_dscales, glm::vec4* dL_drots)
{
	// Recompute (intermediate) results for the 3D covariance computation.
	glm::vec4 q = rot;// / glm::length(rot);
	float r = q.x;
	float x = q.y;
	float y = q.z;
	float z = q.w;

	glm::mat3 R = glm::mat3(
		1.f - 2.f * (y * y + z * z), 2.f * (x * y - r * z), 2.f * (x * z + r * y),
		2.f * (x * y + r * z), 1.f - 2.f * (x * x + z * z), 2.f * (y * z - r * x),
		2.f * (x * z - r * y), 2.f * (y * z + r * x), 1.f - 2.f * (x * x + y * y)
	);

	glm::mat3 S = glm::mat3(1.0f);

	glm::vec3 s = mod * scale;
	S[0][0] = s.x;
	S[1][1] = s.y;
	S[2][2] = s.z;

	glm::mat3 M = S * R;

	const float* dL_dcov3D = dL_dcov3Ds + 6 * idx;

	glm::vec3 dunc(dL_dcov3D[0], dL_dcov3D[3], dL_dcov3D[5]);
	glm::vec3 ounc = 0.5f * glm::vec3(dL_dcov3D[1], dL_dcov3D[2], dL_dcov3D[4]);

	// Convert per-element covariance loss gradients to matrix form
	glm::mat3 dL_dSigma = glm::mat3(
		dL_dcov3D[0], 0.5f * dL_dcov3D[1], 0.5f * dL_dcov3D[2],
		0.5f * dL_dcov3D[1], dL_dcov3D[3], 0.5f * dL_dcov3D[4],
		0.5f * dL_dcov3D[2], 0.5f * dL_dcov3D[4], dL_dcov3D[5]
	);

	// Compute loss gradient w.r.t. matrix M
	// dSigma_dM = 2 * M
	glm::mat3 dL_dM = 2.0f * M * dL_dSigma;

	glm::mat3 Rt = glm::transpose(R);
	glm::mat3 dL_dMt = glm::transpose(dL_dM);

	// Gradients of loss w.r.t. scale
	glm::vec3* dL_dscale = dL_dscales + idx;
	dL_dscale->x = glm::dot(Rt[0], dL_dMt[0]);
	dL_dscale->y = glm::dot(Rt[1], dL_dMt[1]);
	dL_dscale->z = glm::dot(Rt[2], dL_dMt[2]);

	dL_dMt[0] *= s.x;
	dL_dMt[1] *= s.y;
	dL_dMt[2] *= s.z;

	// Gradients of loss w.r.t. normalized quaternion
	glm::vec4 dL_dq;
	dL_dq.x = 2 * z * (dL_dMt[0][1] - dL_dMt[1][0]) + 2 * y * (dL_dMt[2][0] - dL_dMt[0][2]) + 2 * x * (dL_dMt[1][2] - dL_dMt[2][1]);
	dL_dq.y = 2 * y * (dL_dMt[1][0] + dL_dMt[0][1]) + 2 * z * (dL_dMt[2][0] + dL_dMt[0][2]) + 2 * r * (dL_dMt[1][2] - dL_dMt[2][1]) - 4 * x * (dL_dMt[2][2] + dL_dMt[1][1]);
	dL_dq.z = 2 * x * (dL_dMt[1][0] + dL_dMt[0][1]) + 2 * r * (dL_dMt[2][0] - dL_dMt[0][2]) + 2 * z * (dL_dMt[1][2] + dL_dMt[2][1]) - 4 * y * (dL_dMt[2][2] + dL_dMt[0][0]);
	dL_dq.w = 2 * r * (dL_dMt[0][1] - dL_dMt[1][0]) + 2 * x * (dL_dMt[2][0] + dL_dMt[0][2]) + 2 * y * (dL_dMt[1][2] + dL_dMt[2][1]) - 4 * z * (dL_dMt[1][1] + dL_dMt[0][0]);

	// Gradients of loss w.r.t. unnormalized quaternion
	float4* dL_drot = (float4*)(dL_drots + idx);
	*dL_drot = float4{ dL_dq.x, dL_dq.y, dL_dq.z, dL_dq.w };//dnormvdv(float4{ rot.x, rot.y, rot.z, rot.w }, float4{ dL_dq.x, dL_dq.y, dL_dq.z, dL_dq.w });
}

// Backward pass of the preprocessing steps, except
// for the covariance computation and inversion
// (those are handled by a previous kernel call)
template<int C>
__global__ void preprocessCUDA(
	int P, int D, int M,
	const float3* means,
	const int* radii,
	const float* shs,
	const bool* clamped,
	const glm::vec3* scales,
	const glm::vec4* rotations,
	const float scale_modifier,
	const float* view,
	const float* proj,
	const glm::vec3* campos,
	const float3* dL_dmean2D,
	glm::vec3* dL_dmeans,
	float* dL_dcolor,
	const float* dL_ddepth,
	const float2* dL_ddepth_plane,
	float* dL_dcov3D,
	float* dL_dsh,
	glm::vec3* dL_dscale,
	glm::vec4* dL_drot)
{
	auto idx = cg::this_grid().thread_rank();
	if (idx >= P || !(radii[idx] > 0))
		return;

	float3 m = means[idx];

	// Taking care of gradients from the screenspace points
	float4 m_hom = transformPoint4x4(m, proj);
	float m_w = 1.0f / (m_hom.w + 0.0000001f);

	// Compute loss gradient w.r.t. 3D means due to gradients of 2D means
	// from rendering procedure
	glm::vec3 dL_dmean;
	float mul1 = (proj[0] * m.x + proj[4] * m.y + proj[8] * m.z + proj[12]) * m_w * m_w;
	float mul2 = (proj[1] * m.x + proj[5] * m.y + proj[9] * m.z + proj[13]) * m_w * m_w;
	dL_dmean.x = (proj[0] * m_w - proj[3] * mul1) * dL_dmean2D[idx].x + (proj[1] * m_w - proj[3] * mul2) * dL_dmean2D[idx].y;
	dL_dmean.y = (proj[4] * m_w - proj[7] * mul1) * dL_dmean2D[idx].x + (proj[5] * m_w - proj[7] * mul2) * dL_dmean2D[idx].y;
	dL_dmean.z = (proj[8] * m_w - proj[11] * mul1) * dL_dmean2D[idx].x + (proj[9] * m_w - proj[11] * mul2) * dL_dmean2D[idx].y;

	// That's the second part of the mean gradient. Previous computation
	// of cov2D and following SH conversion also affects it.
	dL_dmeans[idx] += dL_dmean;

	// the w must be equal to 1 for view^T * [x,y,z,1]
	float3 m_view = transformPoint4x3(m, view);

	// Compute loss gradient w.r.t. 3D means due to gradients of depth
	// from rendering procedure
	glm::vec3 dL_dmean2;
	float mul3 = view[2] * m.x + view[6] * m.y + view[10] * m.z + view[14];
	dL_dmean2.x = (view[2] - view[3] * mul3) * dL_ddepth[idx];
	dL_dmean2.y = (view[6] - view[7] * mul3) * dL_ddepth[idx];
	dL_dmean2.z = (view[10] - view[11] * mul3) * dL_ddepth[idx];

	// That's the third part of the mean gradient.
	dL_dmeans[idx] += dL_dmean2;

	// Compute gradient updates due to computing colors from SHs
	if (shs)
		computeColorFromSH(idx, D, M, (glm::vec3*)means, *campos, shs, clamped, (glm::vec3*)dL_dcolor, (glm::vec3*)dL_dmeans, (glm::vec3*)dL_dsh);

	// Compute gradient updates due to computing covariance from scale/rotation
	if (scales)
		computeCov3D(idx, scales[idx], scale_modifier, rotations[idx], dL_dcov3D, dL_dscale, dL_drot);
}

// Backward version of the rendering procedure.
template <uint32_t C>
__global__ void __launch_bounds__(BLOCK_X * BLOCK_Y)
renderCUDA(
	const uint2* __restrict__ ranges,
	const uint32_t* __restrict__ point_list,
	int W, int H,
	const float* __restrict__ bg_color,
	const float2* __restrict__ points_xy_image,
	const float4* __restrict__ conic_opacity,
	const float* __restrict__ colors,
	const float* __restrict__ depths,
	const float2* __restrict__ depths_plane,
	const float* __restrict__ alphas,
	const float3* __restrict__ normals,
	const float* __restrict__ wd_map,
	const float* __restrict__ wd2_map,
	const uint32_t* __restrict__ n_contrib,
	const float* __restrict__ dL_dpixels,
	const float* __restrict__ dL_dpixel_depths,
	const float* __restrict__ dL_dpixel_middepths,
	const float* __restrict__ dL_dalphas,
	const float* __restrict__ dL_dpixel_normals,
	const float* __restrict__ dL_ddistortions,
	const float focal_x, 
	const float focal_y,
	float3* __restrict__ dL_dmean2D,
	float4* __restrict__ dL_dconic2D,
	float* __restrict__ dL_dopacity,
	float* __restrict__ dL_dcolors,
	float* __restrict__ dL_ddepths,
	float2* __restrict__ dL_ddepths_plane,
	float3* __restrict__ dL_dnormals
)
{
	// We rasterize again. Compute necessary block info.
	auto block = cg::this_thread_block();
	const uint32_t horizontal_blocks = (W + BLOCK_X - 1) / BLOCK_X;
	const uint2 pix_min = { block.group_index().x * BLOCK_X, block.group_index().y * BLOCK_Y };
	const uint2 pix_max = { min(pix_min.x + BLOCK_X, W), min(pix_min.y + BLOCK_Y , H) };
	const uint2 pix = { pix_min.x + block.thread_index().x, pix_min.y + block.thread_index().y };
	const uint32_t pix_id = W * pix.y + pix.x;
	const float2 pixf = { (float)pix.x, (float)pix.y };

	const bool inside = pix.x < W&& pix.y < H;
	const uint2 range = ranges[block.group_index().y * horizontal_blocks + block.group_index().x];

	const int rounds = ((range.y - range.x + BLOCK_SIZE - 1) / BLOCK_SIZE);

	bool done = !inside;
	int toDo = range.y - range.x;

	__shared__ int collected_id[BLOCK_SIZE];
	__shared__ float2 collected_xy[BLOCK_SIZE];
	__shared__ float4 collected_conic_opacity[BLOCK_SIZE];
	__shared__ float collected_colors[C * BLOCK_SIZE];
	__shared__ float collected_depths[BLOCK_SIZE];
	__shared__ float2 collected_plane[BLOCK_SIZE];
	__shared__ float collected_normals[3 * BLOCK_SIZE];


	// In the forward, we stored the final value for T, the
	// product of all (1 - alpha) factors. 
	const float T_final = inside ? (1 - alphas[pix_id]) : 0;
	const float w_final = inside ? alphas[pix_id] : 0;
	const float wd_final = inside ? wd_map[pix_id] : 0;
	const float wd2_final = inside ? wd2_map[pix_id] : 0;
	
	float T = T_final;
	float w = w_final;
	float wd = wd_final;
	float wd2 = wd2_final;

	// We start from the back. The ID of the last contributing
	// Gaussian is known from each pixel from the forward.
	uint32_t contributor = toDo;
	const int last_contributor = inside ? n_contrib[pix_id] : 0;
	const int max_contributor = inside ? n_contrib[pix_id + H * W] : 0;

	float accum_rec[C] = { 0 };
	float dL_dpixel[C];
	float accum_depth_rec = 0;
	float dL_ddistortion;
	float accum_distortion_rec = 0;
	float dL_dpixel_depth;
	float accum_alpha_rec = 0;
	float dL_dalpha;
	float accum_normal_rec[3] = {0};
	float dL_dpixel_normal[3];
	float dL_dpixel_middepth=0;
	float accum_w = 0;
	float accum_wd = 0;
	float accum_wd2 = 0;
	
	if (inside) {
		for (int i = 0; i < C; i++)
			dL_dpixel[i] = dL_dpixels[i * H * W + pix_id];
		dL_dpixel_depth = dL_dpixel_depths[pix_id];
		dL_dpixel_middepth = dL_dpixel_middepths[pix_id];
		dL_dalpha = dL_dalphas[pix_id];
		dL_ddistortion = dL_ddistortions[pix_id];
		for (int i = 0; i < 3; i++)
			dL_dpixel_normal[i] = dL_dpixel_normals[i * H * W + pix_id];
	}

	float last_alpha = 0;
	float last_color[C] = { 0 };
	float last_depth = 0;
	float last_dL_dw = 0;
	float last_normal[3] = {0};

	// Gradient of pixel coordinate w.r.t. normalized 
	// screen-space viewport corrdinates (-1 to 1)
	const float ddelx_dx = 0.5 * W;
	const float ddely_dy = 0.5 * H;

	// Traverse all Gaussians
	for (int i = 0; i < rounds; i++, toDo -= BLOCK_SIZE)
	{
		// Load auxiliary data into shared memory, start in the BACK
		// and load them in revers order.
		block.sync();
		const int progress = i * BLOCK_SIZE + block.thread_rank();
		if (range.x + progress < range.y)
		{
			const int coll_id = point_list[range.y - progress - 1];
			collected_id[block.thread_rank()] = coll_id;
			collected_xy[block.thread_rank()] = points_xy_image[coll_id];
			collected_conic_opacity[block.thread_rank()] = conic_opacity[coll_id];
			for (int i = 0; i < C; i++)
				collected_colors[i * BLOCK_SIZE + block.thread_rank()] = colors[coll_id * C + i];
			collected_depths[block.thread_rank()] = depths[coll_id];
			float2 depth_plane = depths_plane[coll_id]; 
			collected_plane[block.thread_rank()] = {depth_plane.x/focal_x,depth_plane.y/focal_y};
			{
				collected_normals[0 * BLOCK_SIZE + block.thread_rank()] = normals[coll_id].x;
				collected_normals[1 * BLOCK_SIZE + block.thread_rank()] = normals[coll_id].y;
				collected_normals[2 * BLOCK_SIZE + block.thread_rank()] = normals[coll_id].z;
			}
		}
		block.sync();

		// Iterate over Gaussians
		for (int j = 0; !done && j < min(BLOCK_SIZE, toDo); j++)
		{
			// Keep track of current Gaussian ID. Skip, if this one
			// is behind the last contributor for this pixel.
			contributor--;
			if (contributor >= last_contributor)
				continue;

			// Compute blending values, as before.
			const float2 xy = collected_xy[j];
			const float2 d = { xy.x - pixf.x, xy.y - pixf.y };
			const float4 con_o = collected_conic_opacity[j];

			const float c_d = collected_depths[j];
			const float2 depth_plane = collected_plane[j];
			float depth = c_d + (depth_plane.x * d.x + depth_plane.y * d.y);
			if (depth <= NEAR_PLANE)
				continue;

			float power = -0.5f * (con_o.x * d.x * d.x + con_o.z * d.y * d.y) - con_o.y * d.x * d.y;
			if (power > 0.0f){
				continue;
			}

			

			const float mapped_max_t = (FAR_PLANE * depth - FAR_PLANE * NEAR_PLANE) / ((FAR_PLANE - NEAR_PLANE) * depth);
			const float dmax_t_dd = (FAR_PLANE * NEAR_PLANE) / ((FAR_PLANE - NEAR_PLANE) * depth * depth);

			const float G = exp(power);
			const float alpha = min(0.99f, con_o.w * G);
			// const float alpha = con_o.w * G;
			if (alpha < 1.0f / 255.0f)
				continue;

			T = T / (1.f - alpha);
			const float dchannel_dcolor = alpha * T;
			const float& dpixel_depth_ddepth = dchannel_dcolor;

			// Propagate gradients to per-Gaussian colors and keep
			// gradients w.r.t. alpha (blending factor for a Gaussian/pixel
			// pair).
			float dL_dopa = 0.0f;
			const int global_id = collected_id[j];
			for (int ch = 0; ch < C; ch++)
			{
				const float c = collected_colors[ch * BLOCK_SIZE + j];
				// Update last color (to be used in the next iteration)
				accum_rec[ch] = last_alpha * last_color[ch] + (1.f - last_alpha) * accum_rec[ch];
				last_color[ch] = c;

				const float dL_dchannel = dL_dpixel[ch];
				dL_dopa += (c - accum_rec[ch]) * dL_dchannel;
				// Update the gradients w.r.t. color of the Gaussian. 
				// Atomic, since this pixel is just one of potentially
				// many that were affected by this Gaussian.
				atomicAdd(&(dL_dcolors[global_id * C + ch]), dchannel_dcolor * dL_dchannel);
			}

			for (int ch = 0; ch < 3; ch++)
			{
				const float c = collected_normals[ch * BLOCK_SIZE + j];
				// Update last color (to be used in the next iteration)
				accum_normal_rec[ch] = last_alpha * last_normal[ch] + (1.f - last_alpha) * accum_normal_rec[ch];
				last_normal[ch] = c;

				const float dL_dchannel = dL_dpixel_normal[ch];
				dL_dopa += (c - accum_normal_rec[ch]) * dL_dchannel;
				// Update the gradients w.r.t. color of the Gaussian. 
				// Atomic, since this pixel is just one of potentially
				// many that were affected by this Gaussian.
				switch (ch)
				{
				case 0:
					atomicAdd(&dL_dnormals[global_id].x, dchannel_dcolor * dL_dchannel);
					break;
				case 1:
					atomicAdd(&dL_dnormals[global_id].y, dchannel_dcolor * dL_dchannel);
					break;
				case 2:
					atomicAdd(&dL_dnormals[global_id].z, dchannel_dcolor * dL_dchannel);
					break;
				default:
					break;
				}
			}
			
			// Propagate gradients from pixel depth to opacity

			accum_depth_rec = last_alpha * last_depth + (1.f - last_alpha) * accum_depth_rec;
			last_depth = depth;
			dL_dopa += (depth - accum_depth_rec) * dL_dpixel_depth;
			float dL_ddepth = dpixel_depth_ddepth * dL_dpixel_depth
								+ 2 * dchannel_dcolor * (w_final * mapped_max_t - wd_final) * dL_ddistortion * dmax_t_dd;
			if (contributor == max_contributor-1) {
				dL_ddepth += dL_dpixel_middepth;
			}
			
			atomicAdd(&(dL_ddepths[global_id]), dL_ddepth);
			atomicAdd(&dL_ddepths_plane[global_id].x, dL_ddepth * d.x / focal_x);
			atomicAdd(&dL_ddepths_plane[global_id].y, dL_ddepth * d.y / focal_y);

			// Propagate gradients from pixel alpha (weights_sum) to opacity
			accum_alpha_rec = last_alpha + (1.f - last_alpha) * accum_alpha_rec;
			dL_dopa += (1 - accum_alpha_rec) * dL_dalpha; //- (alpha - accum_alpha_rec) * dL_dalpha;

			
			// const float d2 = mapped_max_t * mapped_max_t;
			// const float& weight = dchannel_dcolor;
			// w -= weight;
			// wd -= weight * mapped_max_t;
			// wd2 -= weight * d2;
			// float dL1_dw = w * d2 + accum_wd2;
			// float dL2_dw = wd2 + accum_w * d2;
			// float dL3_dw = (wd+accum_wd)*(-2*mapped_max_t);
			// float dL_dw = dL1_dw + dL2_dw + dL3_dw;
			// accum_distortion_rec = last_alpha * last_dL_dw + (1.f - last_alpha) * accum_distortion_rec;
			// dL_dopa += (dL_dw - accum_distortion_rec) * dL_ddistortion;
			// last_dL_dw = dL_dw;

			// accum_w += weight;
			// accum_wd += weight * mapped_max_t;
			// accum_wd2 += weight * d2;



			dL_dopa *= T;
			// Update last alpha (to be used in the next iteration)
			last_alpha = alpha;

			// Account for fact that alpha also influences how much of
			// the background color is added if nothing left to blend
			float bg_dot_dpixel = 0;
			for (int i = 0; i < C; i++)
				bg_dot_dpixel += bg_color[i] * dL_dpixel[i];
			dL_dopa += (-T_final / (1.f - alpha)) * bg_dot_dpixel;


			// Helpful reusable temporary variables
			const float dL_dG = con_o.w * dL_dopa;
			const float gdx = G * d.x;
			const float gdy = G * d.y;
			const float dG_ddelx = -gdx * con_o.x - gdy * con_o.y;
			const float dG_ddely = -gdy * con_o.z - gdx * con_o.y;

			// Update gradients w.r.t. 2D mean position of the Gaussian
			atomicAdd(&dL_dmean2D[global_id].x, (dL_dG * dG_ddelx + dL_ddepth * depth_plane.x) * ddelx_dx);
			atomicAdd(&dL_dmean2D[global_id].y, (dL_dG * dG_ddely + dL_ddepth * depth_plane.y) * ddely_dy);
			// fork from GOF https://github.com/autonomousvision/gaussian-opacity-fields
			const float abs_dL_dmean2D = abs(dL_dG * dG_ddelx * ddelx_dx) + abs(dL_dG * dG_ddely * ddely_dy);
            atomicAdd(&dL_dmean2D[global_id].z, abs_dL_dmean2D);
			// Update gradients w.r.t. 2D covariance (2x2 matrix, symmetric)
			atomicAdd(&dL_dconic2D[global_id].x, -0.5f * gdx * d.x * dL_dG);
			atomicAdd(&dL_dconic2D[global_id].y, -0.5f * gdx * d.y * dL_dG);
			atomicAdd(&dL_dconic2D[global_id].w, -0.5f * gdy * d.y * dL_dG);

			// Update gradients w.r.t. opacity of the Gaussian
			atomicAdd(&(dL_dopacity[global_id]), G * dL_dopa);
		}
	}
}

void BACKWARD::preprocess(
	int P, int D, int M,
	const float3* means3D,
	const int* radii,
	const float* shs,
	const bool* clamped,
	const glm::vec3* scales,
	const glm::vec4* rotations,
	const float scale_modifier,
	const float* cov3Ds,
	const float* viewmatrix,
	const float* projmatrix,
	const float focal_x, float focal_y,
	const float tan_fovx, float tan_fovy,
	const glm::vec3* campos,
	const float3* dL_dmean2D,
	const float* dL_dconic,
	glm::vec3* dL_dmean3D,
	float* dL_dcolor,
	const float* dL_ddepth,
	const float2* dL_ddepth_plane,
	const float3* dL_dnormals,
	float* dL_dcov3D,
	float* dL_dsh,
	glm::vec3* dL_dscale,
	glm::vec4* dL_drot,
	const float4* conic_opacity,
	float* dL_dopacity)
{
	// Propagate gradients for the path of 2D conic matrix computation. 
	// Somewhat long, thus it is its own kernel rather than being part of 
	// "preprocess". When done, loss gradient w.r.t. 3D means has been
	// modified and gradient w.r.t. 3D covariance matrix has been computed.	
	computeCov2DCUDA << <(P + 255) / 256, 256 >> > (
		P,
		means3D,
		radii,
		cov3Ds,
		focal_x,
		focal_y,
		tan_fovx,
		tan_fovy,
		viewmatrix,
		dL_dconic,
		dL_ddepth_plane,
		dL_dnormals,
		(float3*)dL_dmean3D,
		dL_dcov3D,
		conic_opacity,
		dL_dopacity);

	// Propagate gradients for remaining steps: finish 3D mean gradients,
	// propagate color gradients to SH (if desireD), propagate 3D covariance
	// matrix gradients to scale and rotation.
	preprocessCUDA<NUM_CHANNELS> << < (P + 255) / 256, 256 >> > (
		P, D, M,
		(float3*)means3D,
		radii,
		shs,
		clamped,
		(glm::vec3*)scales,
		(glm::vec4*)rotations,
		scale_modifier,
		viewmatrix,
		projmatrix,
		campos,
		(float3*)dL_dmean2D,
		(glm::vec3*)dL_dmean3D,
		dL_dcolor,
		dL_ddepth,
		dL_ddepth_plane,
		dL_dcov3D,
		dL_dsh,
		dL_dscale,
		dL_drot);
}

void BACKWARD::render(
	const dim3 grid, const dim3 block,
	const uint2* ranges,
	const uint32_t* point_list,
	int W, int H,
	const float* bg_color,
	const float2* means2D,
	const float4* conic_opacity,
	const float* colors,
	const float* depths,
	const float2* depths_plane,
	const float* alphas,
	const float3* normals,
	const float* wd_map,
	const float* wd2_map,
	const uint32_t* n_contrib,
	const float* dL_dpixels,
	const float* dL_dpixel_depths,
	const float* dL_dpixel_middepths,
	const float* dL_dalphas,
	const float* dL_dpixel_normals,
	const float* dL_ddistortions,
	const float focal_x, 
	const float focal_y,
	float3* dL_dmean2D,
	float4* dL_dconic2D,
	float* dL_dopacity,
	float* dL_dcolors,
	float* dL_ddepths,
	float2* dL_ddepths_plane,
	float3* dL_dnormals)
{
	renderCUDA<NUM_CHANNELS> << <grid, block >> >(
		ranges,
		point_list,
		W, H,
		bg_color,
		means2D,
		conic_opacity,
		colors,
		depths,
		depths_plane,
		alphas,
		normals,
		wd_map,
		wd2_map,
		n_contrib,
		dL_dpixels,
		dL_dpixel_depths,
		dL_dpixel_middepths,
		dL_dalphas,
		dL_dpixel_normals,
		dL_ddistortions,
		focal_x, 
		focal_y,
		dL_dmean2D,
		dL_dconic2D,
		dL_dopacity,
		dL_dcolors,
		dL_ddepths,
		dL_ddepths_plane,
		dL_dnormals
		);
}