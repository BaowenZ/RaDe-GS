#include "hip/hip_runtime.h"
/*
 * Copyright (C) 2023, Inria
 * GRAPHDECO research group, https://team.inria.fr/graphdeco
 * All rights reserved.
 *
 * This software is free for non-commercial, research and evaluation use 
 * under the terms of the LICENSE.md file.
 *
 * For inquiries contact  george.drettakis@inria.fr
 */

#include "forward.h"
#include "auxiliary.h"
#include <hip/hip_cooperative_groups.h>
#include <cooperative_groups/reduce.h>
#include <iostream>
#include <stdio.h>
#include <cmath>
namespace cg = cooperative_groups;

// Forward method for converting the input spherical harmonics
// coefficients of each Gaussian to a simple RGB color.
__device__ glm::vec3 computeColorFromSH(int idx, int deg, int max_coeffs, const glm::vec3* means, glm::vec3 campos, const float* shs, bool* clamped)
{
	// The implementation is loosely based on code for 
	// "Differentiable Point-Based Radiance Fields for 
	// Efficient View Synthesis" by Zhang et al. (2022)
	glm::vec3 pos = means[idx];
	glm::vec3 dir = pos - campos;
	dir = dir / glm::length(dir);

	glm::vec3* sh = ((glm::vec3*)shs) + idx * max_coeffs;
	glm::vec3 result = SH_C0 * sh[0];

	if (deg > 0)
	{
		float x = dir.x;
		float y = dir.y;
		float z = dir.z;
		result = result - SH_C1 * y * sh[1] + SH_C1 * z * sh[2] - SH_C1 * x * sh[3];

		if (deg > 1)
		{
			float xx = x * x, yy = y * y, zz = z * z;
			float xy = x * y, yz = y * z, xz = x * z;
			result = result +
				SH_C2[0] * xy * sh[4] +
				SH_C2[1] * yz * sh[5] +
				SH_C2[2] * (2.0f * zz - xx - yy) * sh[6] +
				SH_C2[3] * xz * sh[7] +
				SH_C2[4] * (xx - yy) * sh[8];

			if (deg > 2)
			{
				result = result +
					SH_C3[0] * y * (3.0f * xx - yy) * sh[9] +
					SH_C3[1] * xy * z * sh[10] +
					SH_C3[2] * y * (4.0f * zz - xx - yy) * sh[11] +
					SH_C3[3] * z * (2.0f * zz - 3.0f * xx - 3.0f * yy) * sh[12] +
					SH_C3[4] * x * (4.0f * zz - xx - yy) * sh[13] +
					SH_C3[5] * z * (xx - yy) * sh[14] +
					SH_C3[6] * x * (xx - 3.0f * yy) * sh[15];
			}
		}
	}
	result += 0.5f;

	// RGB colors are clamped to positive values. If values are
	// clamped, we need to keep track of this for the backward pass.
	clamped[3 * idx + 0] = (result.x < 0);
	clamped[3 * idx + 1] = (result.y < 0);
	clamped[3 * idx + 2] = (result.z < 0);
	return glm::max(result, 0.0f);
}

// Forward version of 2D covariance matrix computation
template<bool INTE = false>
__device__ bool computeCov2D(const float3& mean, float focal_x, float focal_y, float tan_fovx, float tan_fovy, float kernel_size, const float* cov3D, const float* viewmatrix, 
							float* cov2D, float* camera_plane, float3* output_normal, float2* ray_plane, float& coef, float* invraycov3Ds = nullptr)
{
	// The following models the steps outlined by equations 29
	// and 31 in "EWA Splatting" (Zwicker et al., 2002). 
	// Additionally considers aspect / scaling of viewport.
	// Transposes used to account for row-/column-major conventions.
	float3 t = transformPoint4x3(mean, viewmatrix);

	const float limx = 1.3f * tan_fovx;
	const float limy = 1.3f * tan_fovy;
	float txtz = t.x / t.z;
	float tytz = t.y / t.z;
	t.x = min(limx, max(-limx, txtz)) * t.z;
	t.y = min(limy, max(-limy, tytz)) * t.z;
	txtz = t.x / t.z;
	tytz = t.y / t.z;

	glm::mat3 J = glm::mat3(
		focal_x / t.z, 0.0f, -(focal_x * t.x) / (t.z * t.z),
		0.0f, focal_y / t.z, -(focal_y * t.y) / (t.z * t.z),
		0, 0, 0);

	glm::mat3 W = glm::mat3(
		viewmatrix[0], viewmatrix[4], viewmatrix[8],
		viewmatrix[1], viewmatrix[5], viewmatrix[9],
		viewmatrix[2], viewmatrix[6], viewmatrix[10]);

	glm::mat3 T = W * J;

	glm::mat3 Vrk = glm::mat3(
		cov3D[0], cov3D[1], cov3D[2],
		cov3D[1], cov3D[3], cov3D[4],
		cov3D[2], cov3D[4], cov3D[5]);

	glm::mat3 cov = glm::transpose(T) * glm::transpose(Vrk) * T;

	// output[0] = { float(cov[0][0]), float(cov[0][1]), float(cov[1][1]) };
	cov2D[0] = float(cov[0][0]);
	cov2D[1] = float(cov[0][1]);
	cov2D[2] = float(cov[1][1]);
	const float det_0 = max(1e-6, cov[0][0] * cov[1][1] - cov[0][1] * cov[0][1]);
	const float det_1 = max(1e-6, (cov[0][0] + kernel_size) * (cov[1][1] + kernel_size) - cov[0][1] * cov[0][1]);
	coef = sqrt(det_0 / (det_1+1e-6) + 1e-6);
	if (det_0 <= 1e-6 || det_1 <= 1e-6){
		coef = 0.0f;
	}

	// glm::mat3 testm = glm::mat3{
	// 	1,2,3,
	// 	4,5,6,
	// 	7,8,9,
	// };
	// glm::vec3 testv = {1,1,1};
	// glm::vec3 resultm = testm * testv;
	// printf("%f %f %f\n", resultm[0], resultm[1],resultm[2]); 12.000000 15.000000 18.000000

	glm::mat3 Vrk_eigen_vector;
	glm::vec3 Vrk_eigen_value;
	int D = glm_modification::findEigenvaluesSymReal(Vrk,Vrk_eigen_value,Vrk_eigen_vector);

	unsigned int min_id = Vrk_eigen_value[0]>Vrk_eigen_value[1]? (Vrk_eigen_value[1]>Vrk_eigen_value[2]?2:1):(Vrk_eigen_value[0]>Vrk_eigen_value[2]?2:0);

	glm::mat3 Vrk_inv;
	bool well_conditioned = Vrk_eigen_value[min_id]>0.00000001;
	glm::vec3 eigenvector_min;
	if(well_conditioned)
	{
		glm::mat3 diag = glm::mat3( 1/Vrk_eigen_value[0], 0, 0,
									0, 1/Vrk_eigen_value[1], 0,
									0, 0, 1/Vrk_eigen_value[2] );
		Vrk_inv = Vrk_eigen_vector * diag * glm::transpose(Vrk_eigen_vector);
	}
	else
	{
		eigenvector_min = Vrk_eigen_vector[min_id];
		Vrk_inv = glm::outerProduct(eigenvector_min,eigenvector_min);
	}
	
	glm::mat3 cov_cam_inv = glm::transpose(W) * Vrk_inv * W;
	glm::vec3 uvh = {txtz, tytz, 1};
	glm::vec3 uvh_m = cov_cam_inv * uvh;
	glm::vec3 uvh_mn = glm::normalize(uvh_m);

	if(isnan(uvh_mn.x)|| D==0)
	{
		for(int ch = 0; ch < 6; ch++)
			camera_plane[ch] = 0;
		*output_normal = {0,0,0};
		*ray_plane = {0,0};
	}
	else
	{
		float u2 = txtz * txtz;
		float v2 = tytz * tytz;
		float uv = txtz * tytz;

		float l = sqrt(t.x*t.x+t.y*t.y+t.z*t.z);
		glm::mat3 nJ = glm::mat3(
			1 / t.z, 0.0f, -(t.x) / (t.z * t.z),
			0.0f, 1 / t.z, -(t.y) / (t.z * t.z),
			t.x/l, t.y/l, t.z/l);

		glm::mat3 nJ_inv = glm::mat3(
			v2 + 1,	-uv, 		0,
			-uv,	u2 + 1,		0,
			-txtz,	-tytz,		0
		);

		if constexpr (INTE)
		{
			glm::mat3 inv_cov_ray;
			if(well_conditioned)
			{
				float ltz = u2+v2+1;
				glm::mat3 nJ_inv_full = t.z/(u2+v2+1) * \
										glm::mat3(
											v2 + 1,	-uv, 		txtz/l*ltz,
											-uv,	u2 + 1,		tytz/l*ltz,
											-txtz,	-tytz,		1/l*ltz);
				glm::mat3 T2 = W * glm::transpose(nJ_inv_full);
				inv_cov_ray = glm::transpose(T2) * Vrk_inv * T2;
			}
			else
			{
				glm::mat3 T2 = W * nJ;
				glm::mat3 cov_ray = glm::transpose(T2) * Vrk_inv * T2;
				glm::mat3 cov_eigen_vector;
				glm::vec3 cov_eigen_value;
				glm_modification::findEigenvaluesSymReal(cov_ray,cov_eigen_value,cov_eigen_vector);
				unsigned int min_id = cov_eigen_value[0]>cov_eigen_value[1]? (cov_eigen_value[1]>cov_eigen_value[2]?2:1):(cov_eigen_value[0]>cov_eigen_value[2]?2:0);
				float lambda1 = cov_eigen_value[(min_id+1)%3];
				float lambda2 = cov_eigen_value[(min_id+2)%3];
				float lambda3 = cov_eigen_value[min_id];
				glm::mat3 new_cov_eigen_vector = glm::mat3();
				new_cov_eigen_vector[0] = cov_eigen_vector[(min_id+1)%3];
				new_cov_eigen_vector[1] = cov_eigen_vector[(min_id+2)%3];
				new_cov_eigen_vector[2] = cov_eigen_vector[min_id];
				glm::vec3 r3 = glm::vec3(new_cov_eigen_vector[0][2],new_cov_eigen_vector[1][2],new_cov_eigen_vector[2][2]);

				glm::mat3 cov2d = glm::mat3(
					1/lambda1,0,-r3[0]/r3[2]/lambda1,
					0,1/lambda2,-r3[1]/r3[2]/lambda2,
					-r3[0]/r3[2]/lambda1,-r3[1]/r3[2]/lambda2,0
				);
				glm::mat3 inv_cov_ray = new_cov_eigen_vector * cov2d * glm::transpose(new_cov_eigen_vector);
			}
			glm::mat3 scale = glm::mat3(1/focal_x,0,0,
										0, 1/focal_y,0,
										0,0,1);
			inv_cov_ray = scale * inv_cov_ray * scale;
			invraycov3Ds[0] = inv_cov_ray[0][0];
			invraycov3Ds[1] = inv_cov_ray[0][1];
			invraycov3Ds[2] = inv_cov_ray[0][2];
			invraycov3Ds[3] = inv_cov_ray[1][1];
			invraycov3Ds[4] = inv_cov_ray[1][2];
			invraycov3Ds[5] = inv_cov_ray[2][2];
		}


		float vbn = glm::dot(uvh_mn, uvh);
		float factor_normal = l / (u2+v2+1);
		glm::vec3 plane = nJ_inv * (uvh_mn/max(vbn,0.0000001f));
		float nl = u2+v2+1;
		glm::vec2 camera_plane_x = {(-(v2 + 1)*t.z+plane[0]*t.x)/nl/focal_x, (uv*t.z+plane[1]*t.x)/nl/focal_y};
		glm::vec2 camera_plane_y = {(uv*t.z+plane[0]*t.y)/nl/focal_x, (-(u2 + 1)*t.z+plane[1]*t.y)/nl/focal_y};
		glm::vec2 camera_plane_z = {(t.x+plane[0]*t.z)/nl/focal_x, (t.y+plane[1]*t.z)/nl/focal_y};

		*ray_plane = {plane[0]*l/nl/focal_x, plane[1]*l/nl/focal_y};

		camera_plane[0] = camera_plane_x.x;
		camera_plane[1] = camera_plane_x.y;
		camera_plane[2] = camera_plane_y.x;
		camera_plane[3] = camera_plane_y.y;
		camera_plane[4] = camera_plane_z.x;
		camera_plane[5] = camera_plane_z.y;


		glm::vec3 ray_normal_vector = {-plane[0]*factor_normal, -plane[1]*factor_normal, -1};
		glm::vec3 cam_normal_vector = nJ * ray_normal_vector;
		glm::vec3 normal_vector = glm::normalize(cam_normal_vector);

		*output_normal = {normal_vector.x, normal_vector.y, normal_vector.z};

	}
	return well_conditioned;
}


// Forward method for converting scale and rotation properties of each
// Gaussian to a 3D covariance matrix in world space. Also takes care
// of quaternion normalization.
__device__ void computeCov3D(const glm::vec3 scale, float mod, const glm::vec4 rot, float* cov3D)
{
	// Create scaling matrix
	glm::mat3 S = glm::mat3(1.0f);
	S[0][0] = mod * scale.x;
	S[1][1] = mod * scale.y;
	S[2][2] = mod * scale.z;

	// Normalize quaternion to get valid rotation
	glm::vec4 q = rot;// / glm::length(rot);
	float r = q.x;
	float x = q.y;
	float y = q.z;
	float z = q.w;

	// Compute rotation matrix from quaternion
	glm::mat3 R = glm::mat3(
		1.f - 2.f * (y * y + z * z), 2.f * (x * y - r * z), 2.f * (x * z + r * y),
		2.f * (x * y + r * z), 1.f - 2.f * (x * x + z * z), 2.f * (y * z - r * x),
		2.f * (x * z - r * y), 2.f * (y * z + r * x), 1.f - 2.f * (x * x + y * y)
	);

	glm::mat3 M = S * R;

	// Compute 3D world covariance matrix Sigma
	glm::mat3 Sigma = glm::transpose(M) * M;

	// Covariance is symmetric, only store upper right
	cov3D[0] = Sigma[0][0];
	cov3D[1] = Sigma[0][1];
	cov3D[2] = Sigma[0][2];
	cov3D[3] = Sigma[1][1];
	cov3D[4] = Sigma[1][2];
	cov3D[5] = Sigma[2][2];
}

// Perform initial steps for each Gaussian prior to rasterization.
template<int C, bool INTE = false>
__global__ void preprocessCUDA(int P, int D, int M,
	const float* orig_points,
	const glm::vec3* scales,
	const float scale_modifier,
	const glm::vec4* rotations,
	const float* opacities,
	const float* shs,
	bool* clamped,
	const float* cov3D_precomp,
	const float* colors_precomp,
	const float* viewmatrix,
	const float* projmatrix,
	const glm::vec3* cam_pos,
	const int W, int H,
	const float tan_fovx, float tan_fovy,
	const float focal_x, float focal_y,
	const float kernel_size,
	int* radii,
	float2* points_xy_image,
	float3* view_points,
	float* depths,
	float* camera_planes,
	float2* ray_planes,
	float3* normals,
	float* cov3Ds,
	float* rgb,
	float4* conic_opacity,
	const dim3 grid,
	uint32_t* tiles_touched,
	bool prefiltered,
	float* invraycov3Ds,
	float* ts,
	bool* conditions)
{
	auto idx = cg::this_grid().thread_rank();
	if (idx >= P)
		return;

	// Initialize radius and touched tiles to 0. If this isn't changed,
	// this Gaussian will not be processed further.
	radii[idx] = 0;
	tiles_touched[idx] = 0;
	// Perform near culling, quit if outside.
	float3 p_view;
	if (!in_frustum(idx, orig_points, viewmatrix, projmatrix, prefiltered, p_view))
		return;
	// Transform point by projecting
	float3 p_orig = { orig_points[3 * idx], orig_points[3 * idx + 1], orig_points[3 * idx + 2] };
	float4 p_hom = transformPoint4x4(p_orig, projmatrix);
	float p_w = 1.0f / (p_hom.w + 0.0000001f);
	float3 p_proj = { p_hom.x * p_w, p_hom.y * p_w, p_hom.z * p_w };

	// If 3D covariance matrix is precomputed, use it, otherwise compute
	// from scaling and rotation parameters. 
	const float* cov3D;
	if (cov3D_precomp != nullptr)
	{
		cov3D = cov3D_precomp + idx * 6;
	}
	else
	{
		computeCov3D(scales[idx], scale_modifier, rotations[idx], cov3Ds + idx * 6);
		cov3D = cov3Ds + idx * 6;
	}

	// Compute 2D screen-space covariance matrix
	float cov2D[3];
	float ceof;
	bool condition = computeCov2D<INTE>(p_orig, focal_x, focal_y, tan_fovx, tan_fovy, kernel_size, cov3D, viewmatrix, cov2D, camera_planes + idx * 6, normals + idx, ray_planes + idx, ceof, invraycov3Ds + idx * 6);
	if constexpr (INTE)
	{
		conditions[idx] = condition;
	}
	ts[idx] = sqrt(p_view.x*p_view.x+p_view.y*p_view.y+p_view.z*p_view.z);
	const float3 cov = {cov2D[0], cov2D[1], cov2D[2]};
	
	// Invert covariance (EWA algorithm)
	float det = (cov.x * cov.z - cov.y * cov.y);
	if (det == 0.0f)
		return;
	float det_inv = 1.f / det;
	float3 conic = { cov.z * det_inv, -cov.y * det_inv, cov.x * det_inv };

	// Compute extent in screen space (by finding eigenvalues of
	// 2D covariance matrix). Use extent to compute a bounding rectangle
	// of screen-space tiles that this Gaussian overlaps with. Quit if
	// rectangle covers 0 tiles. 
	float mid = 0.5f * (cov.x + cov.z);
	float lambda1 = mid + sqrt(max(0.1f, mid * mid - det));
	float lambda2 = mid - sqrt(max(0.1f, mid * mid - det));
	float my_radius = ceil(3.f * sqrt(max(lambda1, lambda2)));
	float2 point_image = { ndc2Pix(p_proj.x, W), ndc2Pix(p_proj.y, H) };
	uint2 rect_min, rect_max;
	getRect(point_image, my_radius, rect_min, rect_max, grid);
	if ((rect_max.x - rect_min.x) * (rect_max.y - rect_min.y) == 0)
		return;

	// If colors have been precomputed, use them, otherwise convert
	// spherical harmonics coefficients to RGB color.
	if (colors_precomp == nullptr)
	{
		glm::vec3 result = computeColorFromSH(idx, D, M, (glm::vec3*)orig_points, *cam_pos, shs, clamped);
		rgb[idx * C + 0] = result.x;
		rgb[idx * C + 1] = result.y;
		rgb[idx * C + 2] = result.z;
	}

	// Store some useful helper data for the next steps.
	depths[idx] = p_view.z;
	view_points[idx] = p_view;
	radii[idx] = my_radius;
	points_xy_image[idx] = point_image;
	// Inverse 2D covariance and opacity neatly pack into one float4
	conic_opacity[idx] = { conic.x, conic.y, conic.z, opacities[idx] * ceof};
	tiles_touched[idx] = (rect_max.y - rect_min.y) * (rect_max.x - rect_min.x);
}

// Main rasterization method. Collaboratively works on one tile per
// block, each thread treats one pixel. Alternates between fetching 
// and rasterizing data.
template <uint32_t CHANNELS, bool COORD, bool DEPTH, bool NORMAL>
__global__ void __launch_bounds__(BLOCK_X * BLOCK_Y)
renderCUDA(
	const uint2* __restrict__ ranges,
	const uint32_t* __restrict__ point_list,
	int W, int H,
	const float* __restrict__ view_points,
	const float2* __restrict__ points_xy_image,
	const float* __restrict__ features,
	const float* __restrict__ ts,
	const float* __restrict__ camera_planes,
	const float2* __restrict__ ray_planes,
	const float3* __restrict__ normals,
	const float4* __restrict__ conic_opacity,
	const float focal_x, 
	const float focal_y,
	float* __restrict__ out_alpha,
	uint32_t* __restrict__ n_contrib,
	const float* __restrict__ bg_color,
	float* __restrict__ out_color,
	float* __restrict__ out_coord,
	float* __restrict__ out_mcoord,
	float* __restrict__ out_normal,
	float* __restrict__ out_depth,
	float* __restrict__ out_mdepth,
	float* __restrict__ accum_coord,
	float* __restrict__ accum_depth,
	float* __restrict__ normal_length
	)
{
	// Identify current tile and associated min/max pixel range.
	auto block = cg::this_thread_block();
	uint32_t horizontal_blocks = (W + BLOCK_X - 1) / BLOCK_X;
	uint2 pix_min = { block.group_index().x * BLOCK_X, block.group_index().y * BLOCK_Y };
	uint2 pix_max = { min(pix_min.x + BLOCK_X, W), min(pix_min.y + BLOCK_Y , H) };
	uint2 pix = { pix_min.x + block.thread_index().x, pix_min.y + block.thread_index().y };
	uint32_t pix_id = W * pix.y + pix.x;
	float2 pixf = { (float)pix.x, (float)pix.y };
	float2 pixnf = {(pixf.x-W/2.f)/focal_x,(pixf.y-H/2.f)/focal_y};
	float ln = sqrt(pixnf.x*pixnf.x+pixnf.y*pixnf.y+1);

	constexpr bool GEO = DEPTH || COORD || NORMAL;

	// Check if this thread is associated with a valid pixel or outside.
	bool inside = pix.x < W&& pix.y < H;
	// Done threads can help with fetching, but don't rasterize
	bool done = !inside;

	// Load start/end range of IDs to process in bit sorted list.
	uint2 range = ranges[block.group_index().y * horizontal_blocks + block.group_index().x];
	const int rounds = ((range.y - range.x + BLOCK_SIZE - 1) / BLOCK_SIZE);
	int toDo = range.y - range.x;

	// Allocate storage for batches of collectively fetched data.
	__shared__ int collected_id[BLOCK_SIZE];
	__shared__ float2 collected_xy[BLOCK_SIZE];
	__shared__ float collected_feature[BLOCK_SIZE * CHANNELS];
	__shared__ float collected_camera_plane[BLOCK_SIZE * 6];
	__shared__ float collected_mean3d[BLOCK_SIZE * 3];
	__shared__ float4 collected_conic_opacity[BLOCK_SIZE];
	__shared__ float collected_ts[BLOCK_SIZE];
	__shared__ float2 collected_ray_planes[BLOCK_SIZE];
	__shared__ float3 collected_normals[BLOCK_SIZE];

	// Initialize helper variables
	float T = 1.0f;
	uint32_t contributor = 0;
	uint32_t last_contributor = 0;
	uint32_t max_contributor = -1;
	float C[CHANNELS] = { 0 };
	float weight = 0;
	float Coord[3] = { 0 };
	float mCoord[3] = { 0 };
	float Depth = 0;
	float mDepth = 0;
	float Normal[3] = {0};
	float last_depth = 0;
	float last_weight = 0;

	// Iterate over batches until all done or range is complete
	for (int i = 0; i < rounds; i++, toDo -= BLOCK_SIZE)
	{
		// End if entire block votes that it is done rasterizing
		int num_done = __syncthreads_count(done);
		if (num_done == BLOCK_SIZE)
			break;

		// Collectively fetch per-Gaussian data from global to shared
		int progress = i * BLOCK_SIZE + block.thread_rank();
		if (range.x + progress < range.y)
		{
			int coll_id = point_list[range.x + progress];
			collected_id[block.thread_rank()] = coll_id;
			collected_xy[block.thread_rank()] = points_xy_image[coll_id];
			collected_conic_opacity[block.thread_rank()] = conic_opacity[coll_id];
			for(int ch = 0; ch < CHANNELS; ch++)
				collected_feature[ch * BLOCK_SIZE + block.thread_rank()] = features[coll_id * CHANNELS + ch];
			if constexpr (COORD)
			{
				for(int ch = 0; ch < 6; ch++)
					collected_camera_plane[ch * BLOCK_SIZE + block.thread_rank()] = camera_planes[coll_id * 6 + ch];
				for(int ch = 0; ch < 3; ch++)
					collected_mean3d[ch * BLOCK_SIZE + block.thread_rank()] = view_points[coll_id * 3 + ch];
			}
			if constexpr (DEPTH)
			{
				collected_ts[block.thread_rank()] = ts[coll_id];
				collected_ray_planes[block.thread_rank()] = ray_planes[coll_id];
			}
			if constexpr (NORMAL)
			{
				collected_normals[block.thread_rank()] = normals[coll_id];
			}
		}
		block.sync();

		// Iterate over current batch
		for (int j = 0; !done && j < min(BLOCK_SIZE, toDo); j++)
		{
			// Keep track of current position in range
			contributor++;

			// Resample using conic matrix (cf. "Surface 
			// Splatting" by Zwicker et al., 2001)
			float2 xy = collected_xy[j];
			float2 d = { xy.x - pixf.x, xy.y - pixf.y };
			float4 con_o = collected_conic_opacity[j];
			float power = -0.5f * (con_o.x * d.x * d.x + con_o.z * d.y * d.y) - con_o.y * d.x * d.y;
			if (power > 0.0f){
				continue;
			}
				

			// Eq. (2) from 3D Gaussian splatting paper.
			// Obtain alpha by multiplying with Gaussian opacity
			// and its exponential falloff from mean.
			// Avoid numerical instabilities (see paper appendix). 
			float alpha = min(0.99f, con_o.w * exp(power));
			if (alpha < 1.0f / 255.0f)
				continue;
			float test_T = T * (1 - alpha);
			if (test_T < 0.0001f)
			{
				done = true;
				continue;
			}

			const float aT = alpha * T;
			// Eq. (3) from 3D Gaussian splatting paper.
			for (int ch = 0; ch < CHANNELS; ch++)
				C[ch] += collected_feature[j + BLOCK_SIZE * ch] * aT;

			bool before_median = T > 0.5;
			if constexpr (COORD)
			{
				float2 camera_plane0 = {collected_camera_plane[j], collected_camera_plane[j + BLOCK_SIZE]};
				float2 camera_plane1 = {collected_camera_plane[j + BLOCK_SIZE * 2], collected_camera_plane[j + BLOCK_SIZE * 3]};
				float2 camera_plane2 = {collected_camera_plane[j + BLOCK_SIZE * 4], collected_camera_plane[j + BLOCK_SIZE * 5]};
				float coord[3] = {collected_mean3d[j] + camera_plane0.x * d.x + camera_plane0.y * d.y,
									collected_mean3d[j + BLOCK_SIZE] + camera_plane1.x * d.x + camera_plane1.y * d.y,
									collected_mean3d[j + BLOCK_SIZE * 2] + camera_plane2.x * d.x + camera_plane2.y * d.y};
				for(int ch = 0; ch < 3; ch++)
					Coord[ch] += coord[ch] * aT;
				if (before_median){
					for(int ch = 0; ch < 3; ch++)
						mCoord[ch] = coord[ch];
				}
			}
			if constexpr (DEPTH)
			{
				float t_center = collected_ts[j];
				float2 ray_plane = collected_ray_planes[j];
				float t = t_center + (ray_plane.x * d.x + ray_plane.y * d.y);
				// float depth = t/ln;
				Depth += t * aT;
				if (before_median) mDepth = t;
			}

			if constexpr (NORMAL)
			{
				Normal[0] += collected_normals[j].x * aT;
				Normal[1] += collected_normals[j].y * aT;
				Normal[2] += collected_normals[j].z * aT;
			}
			
			if constexpr (GEO)
			{
				if (before_median)
					max_contributor = contributor;
			}

			
			weight += aT;
			T = test_T;

			// Keep track of last range entry to update this
			// pixel.
			last_contributor = contributor;
		}
	}

	// All threads that treat valid pixel write out their final
	// rendering data to the frame and auxiliary buffers.
	if (inside)
	{
		n_contrib[pix_id] = last_contributor;
		n_contrib[pix_id + H * W] = max_contributor;
		for (int ch = 0; ch < CHANNELS; ch++)
			out_color[ch * H * W + pix_id] = C[ch] + T * bg_color[ch];
		out_alpha[pix_id] = weight; //1 - T;

		if constexpr (COORD)
		{
			if(last_contributor)
			{
				for (int ch = 0; ch < 3; ch++)
				{
					out_coord[ch * H * W + pix_id] = Coord[ch] / weight;
				}
			}
			else
			{
				for (int ch = 0; ch < 3; ch++)
					out_coord[ch * H * W + pix_id] = 0;
			}
			for (int ch = 0; ch < 3; ch++)
			{
				accum_coord[ch * H * W + pix_id] = Coord[ch];
				out_mcoord[ch * H * W + pix_id] = mCoord[ch];
			}
		}

		if constexpr (DEPTH)
		{
			float depth_ln = Depth/ln;
			accum_depth[pix_id] = depth_ln;
			if(last_contributor)
			{
				out_depth[pix_id] = depth_ln/weight;
			}
			else
			{
				out_depth[pix_id] = 0;
			}
			out_mdepth[pix_id] = mDepth/ln;
		}

		if constexpr (NORMAL)
		{
			if(last_contributor)
			{
				float len_normal = sqrt(Normal[0]*Normal[0]+Normal[1]*Normal[1]+Normal[2]*Normal[2]);
				normal_length[pix_id] = len_normal;
				len_normal = max(len_normal, NORMALIZE_EPS);
				for (int ch = 0; ch < 3; ch++)
					out_normal[ch * H * W + pix_id] = Normal[ch]/len_normal;
			}
			else
			{
				normal_length[pix_id] = 1;
				for (int ch = 0; ch < 3; ch++)
					out_normal[ch * H * W + pix_id] = 0;
			}
		}
	}
}

// the Bool inputs can be replaced by an enumeration variable for different functions.
void FORWARD::render(
	const dim3 grid, dim3 block,
	const uint2* ranges,
	const uint32_t* point_list,
	int W, int H,
	const float* view_points,
	const float2* means2D,
	const float* colors,
	const float* ts,
	const float* camera_planes,
	const float2* ray_planes,
	const float3* normals,
	const float4* conic_opacity,
	const float focal_x, float focal_y,
	float* out_alpha,
	uint32_t* n_contrib,
	const float* bg_color,
	float* out_color,
	float* out_coord,
	float* out_mcoord,
	float* out_normal,
	float* out_depth,
	float* out_mdepth,
	float* accum_coord,
	float* accum_depth,
	float* normal_length,
	bool require_coord,
	bool require_depth)
{
#define RENDER_CUDA_CALL(template_coord, template_depth, template_normal) \
renderCUDA<NUM_CHANNELS, template_coord, template_depth, template_normal> <<<grid, block>>> ( \
	ranges, point_list, W, H, view_points, means2D, colors, ts, camera_planes, ray_planes, \
	normals, conic_opacity, focal_x, focal_y, out_alpha, n_contrib, bg_color, out_color, \
	out_coord, out_mcoord, out_normal, out_depth, out_mdepth, \
	accum_coord, accum_depth, normal_length)

	if (require_coord && require_depth)
		RENDER_CUDA_CALL(true, true, true);
	else if (require_coord && !require_depth)
		RENDER_CUDA_CALL(true, false, true);
	else if(!require_coord && require_depth)
		RENDER_CUDA_CALL(false, true, true);
	else
		RENDER_CUDA_CALL(false, false, false);
		
#undef RENDER_CUDA_CALL
}

void FORWARD::preprocess(int P, int D, int M,
	const float* means3D,
	const glm::vec3* scales,
	const float scale_modifier,
	const glm::vec4* rotations,
	const float* opacities,
	const float* shs,
	bool* clamped,
	const float* cov3D_precomp,
	const float* colors_precomp,
	const float* viewmatrix,
	const float* projmatrix,
	const glm::vec3* cam_pos,
	const int W, int H,
	const float focal_x, float focal_y,
	const float tan_fovx, float tan_fovy,
	const float kernel_size,
	int* radii,
	float2* means2D,
	float3* view_points,
	float* depths,
	float* camera_planes,
	float2* ray_planes,
	float* ts,
	float3* normals,
	float* cov3Ds,
	float* rgb,
	float4* conic_opacity,
	const dim3 grid,
	uint32_t* tiles_touched,
	bool prefiltered,
	bool integrate,
	float* invraycov3Ds,
	bool* condition)
{
	if(integrate)
		preprocessCUDA<NUM_CHANNELS, true> << <(P + 255) / 256, 256 >> > (
			P, D, M,
			means3D,
			scales,
			scale_modifier,
			rotations,
			opacities,
			shs,
			clamped,
			cov3D_precomp,
			colors_precomp,
			viewmatrix, 
			projmatrix,
			cam_pos,
			W, H,
			tan_fovx, tan_fovy,
			focal_x, focal_y,
			kernel_size,
			radii,
			means2D,
			view_points,
			depths,
			camera_planes,
			ray_planes,
			normals,
			cov3Ds,
			rgb,
			conic_opacity,
			grid,
			tiles_touched,
			prefiltered,
			invraycov3Ds,
			ts,
			condition
			);
	else
		preprocessCUDA<NUM_CHANNELS, false> << <(P + 255) / 256, 256 >> > (
			P, D, M,
			means3D,
			scales,
			scale_modifier,
			rotations,
			opacities,
			shs,
			clamped,
			cov3D_precomp,
			colors_precomp,
			viewmatrix, 
			projmatrix,
			cam_pos,
			W, H,
			tan_fovx, tan_fovy,
			focal_x, focal_y,
			kernel_size,
			radii,
			means2D,
			view_points,
			depths,
			camera_planes,
			ray_planes,
			normals,
			cov3Ds,
			rgb,
			conic_opacity,
			grid,
			tiles_touched,
			prefiltered,
			invraycov3Ds,
			ts,
			condition
			);
}


//follow code is adopted from GOF for marching tetrahedra https://github.com/autonomousvision/gaussian-opacity-fields
// Perform initial steps for each Gaussian prior to rasterization.
template<int C>
__global__ void preprocessPointsCUDA(int P, int D, int M,
	const float* points3D,
	const float* viewmatrix,
	const float* projmatrix,
	const glm::vec3* cam_pos,
	const int W, int H,
	const float tan_fovx, float tan_fovy,
	const float focal_x, float focal_y,
	float2* points2D,
	float* depths,
	const dim3 grid,
	uint32_t* tiles_touched,
	bool prefiltered)
{
	auto idx = cg::this_grid().thread_rank();
	if (idx >= P)
		return;

	// Initialize radius and touched tiles to 0. If this isn't changed,
	// this Gaussian will not be processed further.
	tiles_touched[idx] = 0;

	// Perform near culling, quit if outside.
	float3 p_view;
	if (!in_frustum(idx, points3D, viewmatrix, projmatrix, prefiltered, p_view))
		return;

	// Transform point by projecting
	float3 p_orig = { points3D[3 * idx], points3D[3 * idx + 1], points3D[3 * idx + 2] };
	float4 p_hom = transformPoint4x4(p_orig, projmatrix);
	float p_w = 1.0f / (p_hom.w + 0.0000001f);
	float3 p_proj = { p_hom.x * p_w, p_hom.y * p_w, p_hom.z * p_w };

	float2 point_image = {focal_x * p_view.x / (p_view.z + 0.0000001f) + W/2., focal_y * p_view.y / (p_view.z + 0.0000001f) + H/2.};

	// If the point is outside the image, quit.
	if (point_image.x < 0 || point_image.x >= W || point_image.y < 0 || point_image.y >= H)
		return;

	// Store some useful helper data for the next steps.
	depths[idx] = sqrt(p_view.x*p_view.x+p_view.y*p_view.y+p_view.z*p_view.z);
	points2D[idx] = point_image;
	tiles_touched[idx] = 1;
}

void FORWARD::preprocess_points(int PN, int D, int M,
		const float* points3D,
		const float* viewmatrix,
		const float* projmatrix,
		const glm::vec3* cam_pos,
		const int W, int H,
		const float focal_x, float focal_y,
		const float tan_fovx, float tan_fovy,
		float2* points2D,
		float* depths,
		const dim3 grid,
		uint32_t* tiles_touched,
		bool prefiltered)
{
	preprocessPointsCUDA<NUM_CHANNELS> << <(PN + 255) / 256, 256 >> > (
		PN, D, M,
		points3D,
		viewmatrix, 
		projmatrix,
		cam_pos,
		W, H,
		tan_fovx, tan_fovy,
		focal_x, focal_y,
		points2D,
		depths,
		grid,
		tiles_touched,
		prefiltered
		);
}



// Main rasterization method. Collaboratively works on one tile per
// block, each thread treats one pixel. Alternates between fetching 
// and rasterizing data.
template <uint32_t CHANNELS>
__global__ void __launch_bounds__(BLOCK_X * BLOCK_Y)
integrateCUDA(
	const uint2* __restrict__ gaussian_ranges,
	const uint2* __restrict__ point_ranges,
	const uint32_t* __restrict__ gaussian_list,
	const uint32_t* __restrict__ point_list,
	int W, int H,
	float focal_x, float focal_y,
	const float2* __restrict__ subpixel_offset,
	const float2* __restrict__ points2D,
	const float2* __restrict__ gaussians2D,
	const float* __restrict__ features,
	const float* __restrict__ depths_plane,
	const float2* __restrict__ ray_planes,
	const float* __restrict__ cov3Ds,
	const float* viewmatrix,
	const float3* __restrict__ points3D,
	const float3* __restrict__ gaussians3D,
	const float3* __restrict__ scales,
	const float* __restrict__ invraycov3Ds,
	const float* __restrict__ point_depths,
	const float* __restrict__ gaussian_depths,
	const float4* __restrict__ conic_opacity,
	const bool* __restrict__ condition,
	float* __restrict__ final_T,
	uint32_t* __restrict__ n_contrib,
	// float* __restrict__ center_depth,
	// float4* __restrict__ point_alphas,
	const float* __restrict__ bg_color,
	float* __restrict__ out_color,
	float* __restrict__ out_alpha_integrated,
	float* __restrict__ out_color_integrated,
	float* __restrict__ out_coordinate2d,
	float* __restrict__ out_sdf)
{
	// Identify current tile and associated min/max pixel range.
	auto block = cg::this_thread_block();
	uint32_t horizontal_blocks = (W + BLOCK_X - 1) / BLOCK_X;
	uint2 pix_min = { block.group_index().x * BLOCK_X, block.group_index().y * BLOCK_Y };
	uint2 pix_max = { min(pix_min.x + BLOCK_X, W), min(pix_min.y + BLOCK_Y , H) };
	uint2 pix = { pix_min.x + block.thread_index().x, pix_min.y + block.thread_index().y };
	uint32_t pix_id = W * pix.y + pix.x;
	float2 pixf = { (float)pix.x + 0.5f, (float)pix.y + 0.5f}; // TODO plus 0.5

	// Check if this thread is associated with a valid pixel or outside.
	bool inside = pix.x < W&& pix.y < H;
	// Done threads can help with fetching, but don't rasterize
	bool done = !inside;

	const float depth_input = inside ? subpixel_offset[pix_id].x : 0.0f;

	// Load start/end range of IDs to process in bit sorted list.
	uint2 range = gaussian_ranges[block.group_index().y * horizontal_blocks + block.group_index().x];
	const int rounds = ((range.y - range.x + BLOCK_SIZE - 1) / BLOCK_SIZE);
	int toDo = range.y - range.x;

	uint2 p_range = point_ranges[block.group_index().y * horizontal_blocks + block.group_index().x];
	const int p_rounds = ((p_range.y - p_range.x + BLOCK_SIZE - 1) / BLOCK_SIZE);
	int p_toDo = p_range.y - p_range.x;

	// Allocate storage for batches of collectively fetched data.
	__shared__ int collected_id[BLOCK_SIZE];
	__shared__ float2 collected_xy[BLOCK_SIZE];
	__shared__ float4 collected_conic_opacity[BLOCK_SIZE]; // only need opacity
	__shared__ float collected_depth[BLOCK_SIZE];
	__shared__ float2 collected_depth_plane[BLOCK_SIZE];
	__shared__ float3 collected_scale[BLOCK_SIZE];
	__shared__ float collected_invraycov[BLOCK_SIZE*6];
	__shared__ bool collected_condition[BLOCK_SIZE];
	// Initialize helper variables
	float T = 1.0f;
	uint32_t contributor = 0;
	uint32_t last_contributor = 0;
	float C[CHANNELS*2+2] = { 0 };
	float mid_depth_center = 0;
	float2 mid_plane = {0,0};
	float2 mid_mean2d = {0,0}; 

	uint32_t n_contrib_local = 0;
	uint16_t contributed_ids[MAX_NUM_CONTRIBUTORS*4] = { 0 };
	// use 4 additional corner points so that we have more accurate estimation of contributed_ids
	float corner_Ts[5] = { 1.0f, 1.0f, 1.0f, 1.0f, 1.0f };
	float offset_xs[5] = { 0.0f, -0.5f, 0.5f, -0.5f, 0.5f };
	float offset_ys[5] = { 0.0f, -0.5f, -0.5f, 0.5f, 0.5f };

	// Iterate over batches until all done or range is complete
	for (int i = 0; i < rounds; i++, toDo -= BLOCK_SIZE)
	{
		// End if entire block votes that it is done rasterizing
		int num_done = __syncthreads_count(done);
		if (num_done == BLOCK_SIZE)
			break;

		// Collectively fetch per-Gaussian data from global to shared
		int progress = i * BLOCK_SIZE + block.thread_rank();
		if (range.x + progress < range.y)
		{
			int coll_id = gaussian_list[range.x + progress];
			collected_id[block.thread_rank()] = coll_id;
			collected_conic_opacity[block.thread_rank()] = conic_opacity[coll_id];
			collected_xy[block.thread_rank()] = gaussians2D[coll_id];
			collected_scale[block.thread_rank()] = scales[coll_id];
			collected_depth[block.thread_rank()] = gaussian_depths[coll_id];
			collected_depth_plane[block.thread_rank()] = ray_planes[coll_id];
		}
		block.sync();

		// Iterate over current batch
		for (int j = 0; !done && j < min(BLOCK_SIZE, toDo); j++)
		{
			// Keep track of current position in range
			contributor++;

			// Resample using conic matrix (cf. "Surface 
			// Splatting" by Zwicker et al., 2001)
			// float2 xy = collected_xy[j];
			// float2 d = { xy.x - pixf.x, xy.y - pixf.y };
			float4 con_o = collected_conic_opacity[j];
			float3 scale_j = collected_scale[j];
			float depth_center = collected_depth[j];
			float2 depth_plane = collected_depth_plane[j];
			float2 xy = collected_xy[j];
			bool used = false;
			for (int k = 0; k < 5; ++k){
				float2 d = { xy.x - pixf.x - offset_xs[k], xy.y - pixf.y - offset_ys[k]};
				float depth = depth_center + (depth_plane.x * d.x + depth_plane.y * d.y);
				// depth must be positive otherwise it is not valid and we skip it
				// if (depth <= NEAR_PLANE)
				// 	continue;

				float power = -0.5f * (con_o.x * d.x * d.x + con_o.z * d.y * d.y) - con_o.y * d.x * d.y;
				if (power > 0.0f){
					continue;
				}
				// Eq. (2) from 3D Gaussian splatting paper.
				// Obtain alpha by multiplying with Gaussian opacity
				// and its exponential falloff from mean.
				// Avoid numerical instabilities (see paper appendix). 
				float alpha = min(0.99f, con_o.w * exp(power));
				if (alpha < 1.0f / 255.0f)
					continue;
				float test_T = corner_Ts[k] * (1 - alpha);
				if (test_T < 0.0001f)
				{
					// done = true;
					continue;
				}

				if (k == 0){
					// Eq. (3) from 3D Gaussian splatting paper.
					for (int ch = 0; ch < CHANNELS; ch++)
						C[ch] += features[collected_id[j] * CHANNELS + ch] * alpha * T;
				}
							
				// store maximal depth
				if (depth > C[CHANNELS * 2]){
					C[CHANNELS * 2] = depth;
				}

				if (k == 0){
					C[CHANNELS * 2 + 1] += alpha * T;

					C[CHANNELS] += depth * alpha * T;

					if (T > 0.5){
						C[CHANNELS+1] = depth;
						mid_depth_center = depth_center;
						mid_plane = depth_plane;
						mid_mean2d = xy;
					}

					T = test_T;
				}

				corner_Ts[k] = test_T;
				used = true;

			}

			if (used){
				// Keep track of last range entry to update this
				// pixel.
				last_contributor = contributor;

				contributed_ids[n_contrib_local] = (uint16_t)contributor;
				n_contrib_local += 1;

				if (n_contrib_local >= MAX_NUM_CONTRIBUTORS * 4){
					done = true;
					printf("ERROR: Maximal contributors are met. This should be fixed! %d\n", n_contrib_local);
					break;
				}
			}
		}
	}
	
	// All threads that treat valid pixel write out their final
	// rendering data to the frame and auxiliary buffers.
	if (inside)
	{
		final_T[pix_id] = T;
		n_contrib[pix_id] = last_contributor;

		for (int ch = 0; ch < CHANNELS; ch++)
			out_color[ch * H * W + pix_id] = C[ch] + T * bg_color[ch];
		out_color[3 * H * W + pix_id] = C[CHANNELS];
		out_color[4 * H * W + pix_id] = C[CHANNELS+1];
		// depth and alpha
		out_color[DEPTH_OFFSET * H * W + pix_id] = C[CHANNELS * 2];
		out_color[ALPHA_OFFSET * H * W + pix_id] = C[CHANNELS * 2 + 1];
	}


	// use maximal depth for the current pixel
	const float max_depth = C[CHANNELS * 2];
	
	// Allocate storage for batches of collectively fetched data.
	int projected_ids[MAX_NUM_PROJECTED] = { 0 };
	float2 projected_xy[MAX_NUM_PROJECTED] = { 0.f };
	float projected_depth[MAX_NUM_PROJECTED] = { 0.f };

	//TODO add a for loop here in case we got more points than MAX_NUM_PROJECTED
	uint32_t point_counter_last = 0;
	bool point_done = !inside;
	int total_projected = 0;

	//TODO this for loop is not necessary if we take the minimal value from multiple views
	while (true)
	{
		// End if entire block votes that it is done integrating for all points
		int num_done = __syncthreads_count(point_done);
		if (num_done == BLOCK_SIZE)
			break;

		int num_projected = 0;
		bool excced_max_projected = false;
		done = false;
		
		uint32_t point_counter = 0;
		p_toDo = p_range.y - p_range.x;
		// check how many points projected to this pixel
		// Iterate over batches until all done or range is complete
		for (int i = 0; i < p_rounds; i++, p_toDo -= BLOCK_SIZE)
		{
			//TODO here is not necessary
			// End if entire block votes that it is done rasterizing
			int num_done = __syncthreads_count(done);
			if (num_done == BLOCK_SIZE)
				break;

			block.sync();
			// Collectively fetch per-Gaussian data from global to shared
			int progress = i * BLOCK_SIZE + block.thread_rank();
			if (p_range.x + progress < p_range.y)
			{
				int coll_id = point_list[p_range.x + progress];
				collected_id[block.thread_rank()] = coll_id;
				collected_xy[block.thread_rank()] = points2D[coll_id];
				collected_depth[block.thread_rank()] = point_depths[coll_id];
			}
			block.sync();

			// Iterate over current batch
			for (int j = 0; !done && j < min(BLOCK_SIZE, p_toDo); j++)
			{
				point_counter++;
				if (point_counter <= point_counter_last){
					continue;
				}

				float2 point_xy = collected_xy[j];
				float depth = collected_depth[j];

				// if (abs(point_xy.x - pixf.x) < 0.5 && abs(point_xy.y - pixf.y) < 0.5){
				if ((point_xy.x >= (pixf.x - 0.5)) && (point_xy.x < (pixf.x + 0.5)) && 
					(point_xy.y >= (pixf.y - 0.5)) && (point_xy.y < (pixf.y + 0.5))){
					//TODO check the condition here
					if (true || max_depth <= 0 || depth < max_depth){

						if (num_projected >= MAX_NUM_PROJECTED){
							done = true;
							excced_max_projected = true;
							break;
						}

						projected_ids[num_projected] = collected_id[j];
						projected_xy[num_projected] = point_xy;
						projected_depth[num_projected] = depth;
						num_projected += 1;
					}
				}

			}
		}
		point_counter_last = point_counter - 1;
		point_done = !excced_max_projected;
		total_projected += num_projected;

		// reiterate all primitives
		toDo = range.y - range.x;
		done = false;

		//TODO we could allocate the memory with dynamic size
		float point_alphas[MAX_NUM_PROJECTED] = { 0.f};
		float point_Ts[MAX_NUM_PROJECTED] = {0.f};
		// float point_sdf[MAX_NUM_PROJECTED] = {0.f};
		for (int i = 0; i < num_projected; i++){
			point_Ts[i] = 1.f;
		}


		uint32_t num_iterated = 0;
		bool second_done = !inside;
		uint16_t num_contributed_second = 0;
		//TODO Note that the range is not correct for the near by points, but we use it as approximation for speed up

		// Iterate over batches until all done or range is complete
		for (int i = 0; i < rounds; i++, toDo -= BLOCK_SIZE)
		{
			// End if entire block votes that it is done rasterizing
			int num_done = __syncthreads_count(second_done);
			if (num_done == BLOCK_SIZE)
				break;

			block.sync();
			// Collectively fetch per-Gaussian data from global to shared
			int progress = i * BLOCK_SIZE + block.thread_rank();
			if (range.x + progress < range.y)
			{
				int coll_id = gaussian_list[range.x + progress];
				collected_id[block.thread_rank()] = coll_id;
				collected_conic_opacity[block.thread_rank()] = conic_opacity[coll_id];
				collected_xy[block.thread_rank()] = gaussians2D[coll_id];
				collected_scale[block.thread_rank()] = scales[coll_id];
				collected_depth[block.thread_rank()] = gaussian_depths[coll_id];
				collected_depth_plane[block.thread_rank()] = ray_planes[coll_id];
				collected_condition[block.thread_rank()] = condition[coll_id];
				for (int ii = 0; ii < 6; ii++)
					collected_invraycov[6 * block.thread_rank() + ii] = invraycov3Ds[coll_id * 6 + ii];
			}
			block.sync();

			// Iterate over current batch
			for (int j = 0; !second_done && j < min(BLOCK_SIZE, toDo); j++)
			{
				num_iterated++;
				if (num_iterated > last_contributor){
					second_done = true;
					continue;
				}
				if (num_iterated != (uint32_t)contributed_ids[num_contributed_second]){
					continue;
				} else{
					num_contributed_second += 1;
				}

				float4 con_o = collected_conic_opacity[j];
				float3 scale_j = collected_scale[j];
				float depth_center = collected_depth[j];
				float2 depth_plane = collected_depth_plane[j];
				float2 xy = collected_xy[j];
				glm::mat3 invraycov = glm::mat3(collected_invraycov[6*j+0], collected_invraycov[6*j+1], collected_invraycov[6*j+2],
												collected_invraycov[6*j+1], collected_invraycov[6*j+3], collected_invraycov[6*j+4],
												collected_invraycov[6*j+2], collected_invraycov[6*j+4], collected_invraycov[6*j+5]);
				// iterate over all projected points
				for (int k = 0; k < num_projected; k++){
					// create the ray
					float2 d = { xy.x - projected_xy[k].x, xy.y - projected_xy[k].y};
					float depth = depth_center + (depth_plane.x * d.x + depth_plane.y * d.y);
					float alpha;
					if(collected_condition[j])
					{
						glm::vec3 delta_u = glm::vec3(d.x, d.y, depth_center-min(projected_depth[k],depth));
						float power = -0.5f * (glm::dot(delta_u, invraycov*delta_u));
						alpha = min(0.99f, con_o.w * exp(power));
					}
					else
					{
						if(projected_depth[k]<depth)
							alpha = 0;
						else
						{
							glm::vec3 delta_u = glm::vec3(d.x, d.y, depth_center);
							float power = -0.5f * (glm::dot(delta_u, invraycov*delta_u));
							alpha = min(0.99f, con_o.w * exp(power));
						}
						
					}


					// TODO check here
					if (alpha < 1.0f / 255.0f){
						continue;
					}
						
					float test_T = point_Ts[k] * (1 - alpha);

					point_alphas[k] += alpha * point_Ts[k];

					point_Ts[k] = test_T;
				}
			}
		}
		float3 ray_point = { (pixf.x - W/2.) / focal_x, (pixf.y - H/2.) / focal_y, 1.0 };
		float depth2distance = sqrt(ray_point.x*ray_point.x+ray_point.y*ray_point.y+ray_point.z*ray_point.z);

		// All threads that treat valid pixel write out their final
		// rendering data to the frame and auxiliary buffers.
		if (inside)
		{
			// write alphas
			for (int k = 0; k < num_projected; k++){
				out_alpha_integrated[projected_ids[k]] = point_alphas[k];
				// write colors
				for (int ch = 0; ch < CHANNELS; ch++)
					out_color_integrated[CHANNELS * projected_ids[k] + ch] = C[ch] + T * bg_color[ch];;
				out_coordinate2d[2 * projected_ids[k]] = projected_xy[k].x;
				out_coordinate2d[2 * projected_ids[k]+1] = projected_xy[k].y;
				if(projected_depth[k]>0)
				{
					float2 d = { mid_mean2d.x - projected_xy[k].x, mid_mean2d.y - projected_xy[k].y};
					float depth = mid_depth_center + (mid_plane.x * d.x + mid_plane.y * d.y);
					out_sdf[projected_ids[k]] = depth - projected_depth[k];
				}
			}
		}
	}


	if (inside){
		// use the distortion channel to store the number of projected points
		out_color[DISTORTION_OFFSET * H * W + pix_id] = (float)total_projected; 
	}
}

void FORWARD::integrate(
	const dim3 grid, dim3 block,
	const uint2* gaussian_ranges,
	const uint2* point_ranges,
	const uint32_t* gaussian_list,
	const uint32_t* point_list,
	int W, int H,
	float focal_x, float focal_y,
	const float2* subpixel_offset,
	const float2* points2D,
	const float2* gaussians2D,
	const float* colors,
	const float* depths_plane,
	const float2* ray_planes,
	const float* cov3Ds,
	const float* viewmatrix,
	const float3* points3D,
	const float3* gaussians3D,
	const float3* scales,
	const float* invraycov3Ds,
	const float* point_depths,
	const float* gaussian_depths,
	const float4* conic_opacity,
	const bool* condition,
	float* final_T,
	uint32_t* n_contrib,
	// float* center_depth,
	// float4* center_alphas,
	const float* bg_color,
	float* out_color,
	float* out_alpha_integrated,
	float* out_color_integrated,
	float* out_coordinate2d,
	float* out_sdf)
{
	integrateCUDA<NUM_CHANNELS> << <grid, block >> > (
		gaussian_ranges,
		point_ranges,
		gaussian_list,
		point_list,
		W, H,
		focal_x, focal_y,
		subpixel_offset,
		points2D,
		gaussians2D,
		colors,
		depths_plane,
		ray_planes,
		cov3Ds,
		viewmatrix,
		points3D,
		gaussians3D,
		scales,
		invraycov3Ds,
		point_depths,
		gaussian_depths,
		conic_opacity,
		condition,
		final_T,
		n_contrib,
		// center_depth,
		// center_alphas,
		bg_color,
		out_color,
		out_alpha_integrated,
		out_color_integrated,
		out_coordinate2d,
		out_sdf);
}
